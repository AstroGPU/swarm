#include "hip/hip_runtime.h"
/*************************************************************************
 * Copyright (C) 2011 by Saleh Dindar and the Swarm-NG Development Team  *
 *                                                                       *
 * This program is free software; you can redistribute it and/or modify  *
 * it under the terms of the GNU General Public License as published by  *
 * the Free Software Foundation; either version 3 of the License.        *
 *                                                                       *
 * This program is distributed in the hope that it will be useful,       *
 * but WITHOUT ANY WARRANTY; without even the implied warranty of        *
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the         *
 * GNU General Public License for more details.                          *
 *                                                                       *
 * You should have received a copy of the GNU General Public License     *
 * along with this program; if not, write to the                         *
 * Free Software Foundation, Inc.,                                       *
 * 59 Temple Place - Suite 330, Boston, MA  02111-1307, USA.             *
 ************************************************************************/

#include "swarm/bppt.hpp"
#include "swarm/helpers.hpp"
#include "swarm/gravitation.hpp"
#include "stoppers/stop_on_ejection.hpp"


namespace swarm {

namespace gpu {
namespace bppt {

template< template<class L> class Stopper >
class euler: public integrator {
	typedef integrator base;
	typedef  typename Stopper<gpulog::device_log>::params stop_params_t;
	private:
	double _time_step;
	int _iteration_count;
	stop_params_t _stop_params;

	public:
	euler(const config& cfg): base(cfg),_time_step(0.001), _stop_params(cfg) {
		if(!cfg.count("time step")) ERROR("Integrator gpu_euler requires a timestep ('time step' keyword in the config file).");
		_time_step = atof(cfg.at("time step").c_str());
	}

	virtual void launch_integrator() {
		_iteration_count = _destination_time / _time_step;
		launch_templatized_integrator(this);
	}


	template<class T>
	__device__ void kernel(T a){

		if(sysid()>=_dens.nsys()) return;
		// References to Ensemble and Shared Memory
		ensemble::SystemRef sys = _dens[sysid()];
		typedef typename Gravitation<T::n>::shared_data grav_t;
		Gravitation<T::n> calcForces(sys,*( (grav_t*) system_shared_data_pointer(a) ) );

		// Local variables
		const int nbod = T::n;
		// Body number
		int b = thread_body_idx(nbod);
		// Component number
		int c = thread_component_idx(nbod);
		int ij = thread_in_system();
		bool body_component_grid = (b < nbod) && (c < 3);
		bool first_thread_in_system = thread_in_system() == 0;


		// local variables
		Stopper<gpulog::device_log> stoptest(_stop_params,sys,*_log) ;


		// local information per component per body
		double pos = 0, vel = 0 , acc = 0, jerk = 0;
		if( body_component_grid )
			pos = sys[b][c].pos() , vel = sys[b][c].vel();


		////////// INTEGRATION //////////////////////

		for(int iter = 0 ; (iter < _iteration_count) && sys.active() ; iter ++ ) {
			double h = _time_step;

			calcForces(ij,b,c,pos,vel,acc,jerk);
			// Integratore
			pos = pos +  h*(vel+(h*0.5)*(acc+(h/3.)*jerk));
			vel = vel +  h*(acc+(h*0.5)*jerk);

			// Finalize the step
			if( body_component_grid )
				sys[b][c].pos() = pos , sys[b][c].vel() = vel;
			if( first_thread_in_system ) 
				sys.time() += h;

			if( first_thread_in_system ) 
				sys.active() = ! stoptest() ;

			__syncthreads();


		}

	}


};

/*!
 * \brief Factory to create double/single/mixed euler gpu integrator based on precision
 *
 * @param[in] cfg configuration class
 *
 * @return        pointer to integrator cast to integrator*
 */
extern "C" integrator *create_euler(const config &cfg)
{
	return new euler< stop_on_ejection >(cfg);
}

integrator_plugin_initializer< euler< stop_on_ejection > >
	euler_plugin("euler");

}
}
}
