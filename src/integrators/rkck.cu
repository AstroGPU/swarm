#include "hip/hip_runtime.h"
/*************************************************************************
 * Copyright (C) 2011 by Saleh Dindar and the Swarm-NG Development Team  *
 *                                                                       *
 * This program is free software; you can redistribute it and/or modify  *
 * it under the terms of the GNU General Public License as published by  *
 * the Free Software Foundation; either version 3 of the License.        *
 *                                                                       *
 * This program is distributed in the hope that it will be useful,       *
 * but WITHOUT ANY WARRANTY; without even the implied warranty of        *
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the         *
 * GNU General Public License for more details.                          *
 *                                                                       *
 * You should have received a copy of the GNU General Public License     *
 * along with this program; if not, write to the                         *
 * Free Software Foundation, Inc.,                                       *
 * 59 Temple Place - Suite 330, Boston, MA  02111-1307, USA.             *
 ************************************************************************/

#include "swarm/common.hpp"
#include "swarm/gpu/bppt.hpp"
#include "monitors/composites.hpp"
#include "monitors/stop_on_ejection.hpp"
#include "monitors/log_time_interval.hpp"

namespace swarm { namespace gpu { namespace bppt {

struct FixedTimeStep {
	const static bool adaptive_time_step = false;
	const static bool conditional_accept_step = false;
};

struct AdaptiveTimeStep {
	const static bool adaptive_time_step = true;
	const static bool conditional_accept_step = true;
};

/*! Runge Kutta Cash Karp integrator Fixed/Adaptive
 *
 * \ingroup integrators
 *
 *  This integrator comes in two flavors: Fixed time step, Adaptive time step
 *
 *
 */
template< class AdaptationStyle, class Monitor >
class rkck: public integrator {
	typedef integrator base;
	typedef  Monitor monitor_t;
	typedef  typename monitor_t::params mon_params_t;
	private:
	double _min_time_step;
	double _max_time_step;
	double _error_tolerance;
	int _iteration_count;
	mon_params_t _mon_params;

	public:
	rkck(const config& cfg): base(cfg),_min_time_step(0.001),_max_time_step(0.1), _mon_params(cfg) {
		if(!cfg.count("min_time_step")) ERROR("Integrator rkck requires a min timestep ('min time step' keyword in the config file).");
		_min_time_step = atof(cfg.at("min_time_step").c_str());
		if(!cfg.count("max_time_step")) ERROR("Integrator rkck requires a max timestep ('max time step' keyword in the config file).");
		_max_time_step = atof(cfg.at("max_time_step").c_str());

		if(!cfg.count("error_tolerance")) ERROR("Integrator rkck requires a error tolerance ('error tolerance' keyword in the config file).");
		_error_tolerance = atof(cfg.at("error_tolerance").c_str());
	}

	virtual void launch_integrator() {
		_iteration_count = _destination_time / _max_time_step;
		launch_templatized_integrator(this);
	}


        GPUAPI void convert_internal_to_std_coord() {} 
        GPUAPI void convert_std_to_internal_coord() {}

	GPUAPI bool is_in_body_component_grid(const int b, const int c, const int nbod) 
	{ return ((b < nbod) && (c < 3)); }

	template<class T>
	__device__ void kernel(T compile_time_param){

////////////////////// RKCK Constants /////////////////////////////
	// Cash-Karp constants From GSL
	// Step 1 coefficient
	const double b1 = 1.0 / 5.0;
	// Step 2 coefficient
	const double b2[]  = { 3.0 / 40.0, 9.0 / 40.0 };
	// Step 3 coefficient
	const double b3[]  = { 0.3, -0.9, 1.2 };
	// Step 4 coefficient
	const double b4[]  = { -11.0 / 54.0, 2.5, -70.0 / 27.0, 35.0 / 27.0 };
	// Step 5 coefficient
	const double b5[]  = { 1631.0 / 55296.0, 175.0 / 512.0, 575.0 / 13824.0, 44275.0 / 110592.0, 253.0 / 4096.0 };
	// Step 6 coefficient
	const double b6[]  = { 37.0 / 378.0, 0, 250.0 / 621.0, 125.0 / 594.0, 0 , 512.0 / 1771.0 } ;
	// Error estimation coefficients
	const double ecc[] = { 37.0 / 378.0 - 2825.0 / 27648.0, 0.0, 250.0 / 621.0 - 18575.0 / 48384.0, 125.0 / 594.0 - 13525.0 / 55296.0, -277.00 / 14336.0, 512.0 / 1771.0 - 0.25 };

		if(sysid()>=_dens.nsys()) return;
		// References to Ensemble and Shared Memory
		ensemble::SystemRef sys = _dens[sysid()];
		typedef typename GravitationAccOnly<T::n>::shared_data grav_t;
		GravitationAccOnly<T::n> calcForces(sys,*( (grav_t*) system_shared_data_pointer(this,compile_time_param) ) );

		// Local variables
		const int nbod = T::n;
		// Body number
		const int b = thread_body_idx(nbod);
		// Component number
		const int c = thread_component_idx(nbod);

		// local variables
		monitor_t montest(_mon_params,sys,*_log) ;

		// NB: We use the same shared memory for two purpose and overwrite each other
		// Since the use of the memory is not interleaved, we can safely use the same
		// space for both purposes
		typedef DoubleCoalescedStruct<> shared_mag_t[2][nbod][3];
		shared_mag_t& shared_mag = * (shared_mag_t*) system_shared_data_pointer(this,compile_time_param) ;

		double time_step = _max_time_step;

		// local information per component per body
		double pos = 0, vel = 0 ;
		if( is_in_body_component_grid(b,c,nbod) )
			pos = sys[b][c].pos() , vel = sys[b][c].vel();

   		montest( thread_in_system() );  
		////////// INTEGRATION //////////////////////

		for(int iter = 0 ; (iter < _iteration_count) && sys.is_active() ; iter ++ ) {

			double h = time_step;

			if( sys.time() + h > _destination_time ) {
				h = _destination_time - sys.time();
			}

			//// RKCK   integrate system  ////////////////////////////////////////////////////////////////
			double p0 = pos, v0 = vel;

			// Step 1
			double k1_acc = calcForces.acc(thread_in_system(),b,c,p0,v0);
			double k1_vel = v0;

			double p1 = pos + h * b1 * k1_vel;
			double v1 = vel + h * b1 * k1_acc;

			// Step 2
			double k2_acc = calcForces.acc(thread_in_system(),b,c,p1,v1);
			double k2_vel = v1;

			double p2 = pos + h * ( b2[0] * k1_vel + b2[1] * k2_vel );
			double v2 = vel + h * ( b2[0] * k1_acc + b2[1] * k2_acc );

			// Step 3
			double k3_acc = calcForces.acc(thread_in_system(),b,c,p2,v2);
			double k3_vel = v2;

			double p3 = pos + h * ( b3[0] * k1_vel + b3[1] * k2_vel + b3[2] * k3_vel );
			double v3 = vel + h * ( b3[0] * k1_acc + b3[1] * k2_acc + b3[2] * k3_acc );

			// Step 4
			double k4_acc = calcForces.acc(thread_in_system(),b,c,p3,v3);
			double k4_vel = v3;

			double p4 = pos + h * ( b4[0] * k1_vel + b4[1] * k2_vel + b4[2] * k3_vel + b4[3] * k4_vel );
			double v4 = vel + h * ( b4[0] * k1_acc + b4[1] * k2_acc + b4[2] * k3_acc + b4[3] * k4_acc );

			// Step 5
			double k5_acc = calcForces.acc(thread_in_system(),b,c,p4,v4);
			double k5_vel = v4;

			double p5 = pos + h * ( b5[0] * k1_vel + b5[1] * k2_vel + b5[2] * k3_vel + b5[3] * k4_vel + b5[4] * k5_vel );
			double v5 = vel + h * ( b5[0] * k1_acc + b5[1] * k2_acc + b5[2] * k3_acc + b5[3] * k4_acc + b5[4] * k5_acc );

			// Step 6
			double k6_acc = calcForces.acc(thread_in_system(),b,c,p5,v5);
			double k6_vel = v5;

			double p6 = pos + h * ( b6[0] * k1_vel + b6[1] * k2_vel + b6[2] * k3_vel + b6[3] * k4_vel + b6[4] * k5_vel + b6[5] * k6_vel );
			double v6 = vel + h * ( b6[0] * k1_acc + b6[1] * k2_acc + b6[2] * k3_acc + b6[3] * k4_acc + b6[4] * k5_acc + b6[5] * k6_acc );


			// Error estimate
			double pos_error = h * ( ecc[0] * k1_vel + ecc[1] * k2_vel + ecc[2] * k3_vel + ecc[3] * k4_vel + ecc[4] * k5_vel + ecc[5] * k6_vel );
			double vel_error = h * ( ecc[0] * k1_acc + ecc[1] * k2_acc + ecc[2] * k3_acc + ecc[3] * k4_acc + ecc[4] * k5_acc + ecc[5] * k6_acc );


			bool accept_step = true;

			if( AdaptationStyle::adaptive_time_step ) {
				////////////////////////  Adapting Time step algorithm /////////////////////////////
				const int   integrator_order = 5;
				//! Value used as power in formula to produce larger time step
				const float step_grow_power = -1./(integrator_order+1.);
				//! Value used as power in formula to produce smaller time step
				const float step_shrink_power = -1./integrator_order;
				//! Safety factor to prevent extreme changes in time step
				const float step_guess_safety_factor = 0.9;
				//! Maximum growth of step size allowed at a time
				const float step_grow_max_factor = 5.0; 
				//! Maximum shrinkage of step size allowed at a time
				const float step_shrink_min_factor = 0.2; 

				//  Calculate the error estimate
				if( is_in_body_component_grid(b,c,nbod) ) {

					sys[b][c].pos() = p6 * p6 , sys[b][c].vel() = v6 * v6;
					shared_mag[0][b][c].value() = pos_error * pos_error;
					shared_mag[1][b][c].value() = vel_error * vel_error;
					}
				__syncthreads();

				if( is_in_body_component_grid(b,c,nbod) ) {					// TODO: Could compute the normalized error using one thread per body and reduction (or atomic max)
					if ( (c == 0) && (b == 0) ) {

						double max_error = 0;
						for(int i = 0; i < nbod ; i++){
							double pos_error_mag = shared_mag[0][i][0].value() + shared_mag[0][i][1].value() + shared_mag[0][i][2].value();
							double pos_mag = sys[i][0].pos() + sys[i][1].pos() + sys[i][2].pos();
							double pe = pos_error_mag / pos_mag ;

							double vel_error_mag = shared_mag[1][i][0].value() + shared_mag[1][i][1].value() + shared_mag[1][i][2].value();
							double vel_mag = sys[i][0].vel() + sys[i][1].vel() + sys[i][2].vel();
							double ve = vel_error_mag / vel_mag ;

							max_error = max ( max( pe, ve) , max_error );
						}

						double normalized_error = max_error / _error_tolerance;

						// Calculate New time_step
						double step_guess_power = (normalized_error<1.) ? step_grow_power : step_shrink_power;

						/// factor of 0.5 below due to use of squares in calculate_normalized_error, should we change to match gsl?
						/// gsl uses 1.1, but that seems dangerous, any reason we shouldn't use 1?
						double step_change_factor = ((normalized_error<0.5)||(normalized_error>1.0)) ? step_guess_safety_factor*pow(normalized_error,0.5*step_guess_power) : 1.0;


						//// Update the time step
						double new_time_step = (normalized_error>1.) ? max( time_step * max(step_change_factor,step_shrink_min_factor), _min_time_step ) 
							: min( time_step * max(min(step_change_factor,step_grow_max_factor),1.0), _max_time_step );

						bool accept = ( normalized_error < 1.0 ) || (abs(time_step - new_time_step) < 1e-10) ;

						shared_mag[0][0][0].value() = accept ? 0.0 : 1.0;
						shared_mag[0][0][1].value() = new_time_step;
					}

				}
				__syncthreads();

				time_step = shared_mag[0][0][1].value();
				accept_step = AdaptationStyle::conditional_accept_step ? (shared_mag[0][0][0].value() == 0.0) : true;
				////////////////////////// End of Adaptive time step algorithm  ////////////////////////////////////////////
			}


			if ( accept_step ) {
				// Set the new positions and velocities and time
				pos = p6;
				vel = v6;

				// Finalize the step
				if( is_in_body_component_grid(b,c,nbod) )
					sys[b][c].pos() = pos , sys[b][c].vel() = vel;
				if( (thread_in_system() == 0) ) 
					sys.time() += h;

				montest(thread_in_system());
				if( (thread_in_system() == 0) && sys.is_active() )  {
					if( sys.time() >= _destination_time ) 
					{ sys.set_inactive(); }
				}
			}

			__syncthreads();

		}

	}


};

typedef gpulog::device_log L;
using namespace monitors;

integrator_plugin_initializer<
		rkck< AdaptiveTimeStep, stop_on_ejection<L> >
	> rkck_adaptive_plugin("rkck_adaptive");

integrator_plugin_initializer<
		rkck< FixedTimeStep, stop_on_ejection<L> >
	> rkck_fixed_plugin("rkck_fixed");

integrator_plugin_initializer<
	        rkck< FixedTimeStep, stop_on_ejection_or_close_encounter<L> > 
	> rkck_adaptive_close_encounter_plugin("rkck_adaptive_close_encounter");

integrator_plugin_initializer<
	        rkck< AdaptiveTimeStep, stop_on_ejection_or_close_encounter<L> > 
	> rkck_fixed_close_encounter_plugin("rkck_fixed_close_encounter");

} } } // end namespace bppt :: integrators :: swarm
