#include "hip/hip_runtime.h"
/*************************************************************************
 * Copyright (C) 2011 by Saleh Dindar and the Swarm-NG Development Team  *
 *                                                                       *
 * This program is free software; you can redistribute it and/or modify  *
 * it under the terms of the GNU General Public License as published by  *
 * the Free Software Foundation; either version 3 of the License.        *
 *                                                                       *
 * This program is distributed in the hope that it will be useful,       *
 * but WITHOUT ANY WARRANTY; without even the implied warranty of        *
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the         *
 * GNU General Public License for more details.                          *
 *                                                                       *
 * You should have received a copy of the GNU General Public License     *
 * along with this program; if not, write to the                         *
 * Free Software Foundation, Inc.,                                       *
 * 59 Temple Place - Suite 330, Boston, MA  02111-1307, USA.             *
 ************************************************************************/

#include "swarm/common.hpp"
#include "swarm/gpu/bppt.hpp"
#include "monitors/stop_on_ejection.hpp"
#include "monitors/stop_on_any_large_distance_or_close_encounter.hpp"
#include "monitors/log_time_interval.hpp"
#include "monitors/combine.hpp"


namespace swarm {

namespace gpu {
namespace bppt {

/*! GPU implementation of PEC2 Hermite integrator
 * \ingroup integrators
 *
 */
template< template<class L> class Monitor >
class hermite: public integrator {
	typedef integrator base;
	typedef Monitor<gpulog::device_log> monitor_t;
	typedef typename monitor_t::params mon_params_t;
	private:
	double _time_step;
	mon_params_t _mon_params;

	public:
	hermite(const config& cfg): base(cfg),_time_step(0.001), _mon_params(cfg) {
		_time_step =  cfg.require("time_step", 0.0);
	}

	virtual void launch_integrator() {
		launch_templatized_integrator(this);
	}


	template<class T>
	__device__ void kernel(T compile_time_param){

		if(sysid()>=_dens.nsys()) return;
		// References to Ensemble and Shared Memory
		ensemble::SystemRef sys = _dens[sysid()];
		typedef typename Gravitation<T::n>::shared_data grav_t;
//		Gravitation<T::n> calcForces(sys,*( (grav_t*) system_shared_data_pointer(compile_time_param) ) );
		Gravitation<T::n> calcForces(sys,sysid_in_block());

		// Local variables
		const int nbod = T::n;
		// Body number
		int b = thread_body_idx(nbod);
		// Component number
		int c = thread_component_idx(nbod);
		int ij = thread_in_system();
		bool body_component_grid = (b < nbod) && (c < 3);
		bool first_thread_in_system = thread_in_system() == 0;


		// local variables
		monitor_t montest(_mon_params,sys,*_log) ;


		// local information per component per body
		double pos = 0.0, vel = 0.0 , acc0 = 0.0, jerk0 = 0.0;
		if( body_component_grid )
			pos = sys[b][c].pos() , vel = sys[b][c].vel();


		////////// INTEGRATION //////////////////////

		// Calculate acceleration and jerk
		calcForces(ij,b,c,pos,vel,acc0,jerk0);

		for(int iter = 0 ; (iter < _max_iterations) && sys.is_active() ; iter ++ ) {
			double h = _time_step;

			if( sys.time() + h > _destination_time ) {
				h = _destination_time - sys.time();
			}

			
			// Initial Evaluation
			///calcForces(ij,b,c,pos,vel,acc0,jerk0);

			// Predict 
			pos = pos +  h*(vel+(h*0.5)*(acc0+(h/3.0)*jerk0));
			vel = vel +  h*(acc0+(h*0.5)*jerk0);

			double pre_pos = pos, pre_vel = vel;

			double acc1,jerk1;
			{
				// Evaluation
				calcForces(ij,b,c,pos,vel,acc1,jerk1);
				
				// Correct
				pos = pre_pos + (0.1-0.25) * (acc0 - acc1) * h * h - 1.0/60.0 * ( 7.0 * jerk0 + 2.0 * jerk1 ) * h * h * h;
				vel = pre_vel + ( -0.5 ) * (acc0 - acc1 ) * h -  1.0/12.0 * ( 5.0 * jerk0 + jerk1 ) * h * h;
				//	TODO: Need to test w/ new expressions below
				//				pos = pre_pos + ( (0.1-0.25) * (acc0 - acc1) - 1.0/60.0 * ( 7.0 * jerk0 + 2.0 * jerk1 ) * h) * h * h;
				// vel = pre_vel + (( -0.5 ) * (acc0 - acc1 ) -  1.0/12.0 * ( 5.0 * jerk0 + jerk1 ) * h )* h ;
			}
			{
				// Evaluation
				calcForces(ij,b,c,pos,vel,acc1,jerk1);
				
				// Correct
				pos = pre_pos + (0.1-0.25) * (acc0 - acc1) * h * h - 1.0/60.0 * ( 7.0 * jerk0 + 2.0 * jerk1 ) * h * h * h;
				vel = pre_vel + ( -0.5 ) * (acc0 - acc1 ) * h -  1.0/12.0 * ( 5.0 * jerk0 + jerk1 ) * h * h;
				//	TODO: Need to test w/ new expressions below
				// pos = pre_pos + ((0.1-0.25) * (acc0 - acc1) - 1.0/60.0 * ( 7.0 * jerk0 + 2.0 * jerk1 ) * h )* h * h ;
				// vel = pre_vel + (( -0.5 ) * (acc0 - acc1 ) -  1.0/12.0 * ( 5.0 * jerk0 + jerk1 ) * h ) * h ;
			}
			acc0 = acc1, jerk0 = jerk1;

			// Finalize the step
			if( body_component_grid )
				sys[b][c].pos() = pos , sys[b][c].vel() = vel;
			if( first_thread_in_system ) 
				sys.time() += h;

			if( first_thread_in_system  )  {
			    montest();
			    if( sys.time() >= _destination_time ) 
				sys.set_inactive();
			}

			__syncthreads();


		}

	}


};


// WARNING: EBF: commented out to test new stopper
//integrator_plugin_initializer<hermite< stop_on_ejection > >
//	hermite_plugin("hermite");

integrator_plugin_initializer<hermite< monitors::stop_on_any_large_distance_or_close_encounter > >
	hermite_plugin("hermite");

integrator_plugin_initializer<hermite< monitors::log_time_interval > >
	hermite_log_plugin("hermite_log");


}
}
}
