#include "hip/hip_runtime.h"
/*************************************************************************
 * Copyright (C) 2011 by Saleh Dindar and the Swarm-NG Development Team  *
 *                                                                       *
 * This program is free software; you can redistribute it and/or modify  *
 * it under the terms of the GNU General Public License as published by  *
 * the Free Software Foundation; either version 3 of the License.        *
 *                                                                       *
 * This program is distributed in the hope that it will be useful,       *
 * but WITHOUT ANY WARRANTY; without even the implied warranty of        *
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the         *
 * GNU General Public License for more details.                          *
 *                                                                       *
 * You should have received a copy of the GNU General Public License     *
 * along with this program; if not, write to the                         *
 * Free Software Foundation, Inc.,                                       *
 * 59 Temple Place - Suite 330, Boston, MA  02111-1307, USA.             *
 ************************************************************************/

#include "swarm/common.hpp"
#include "swarm/gpu/bppt.hpp"
#include "monitors/composites.hpp"
#include "monitors/stop_on_ejection.hpp"
#include "monitors/log_time_interval.hpp"


namespace swarm { namespace gpu { namespace bppt {

/*! GPU implementation of PEC2 Hermite integrator
 * \ingroup integrators
 *
 */
template< class Monitor >
class hermite: public integrator {
	typedef integrator base;
	typedef Monitor monitor_t;
	typedef typename monitor_t::params mon_params_t;
	private:
	double _time_step;
	mon_params_t _mon_params;

	public:
	hermite(const config& cfg): base(cfg),_time_step(0.001), _mon_params(cfg) {
		_time_step =  cfg.require("time_step", 0.0);
	}

	virtual void launch_integrator() {
		launch_templatized_integrator(this);
	}


        GPUAPI void convert_internal_to_std_coord() {} 
        GPUAPI void convert_std_to_internal_coord() {}

	template<class T>
	__device__ void kernel(T compile_time_param){

		if(sysid()>=_dens.nsys()) return;
		// References to Ensemble and Shared Memory
		ensemble::SystemRef sys = _dens[sysid()];
		typedef typename Gravitation<T::n>::shared_data grav_t;
		Gravitation<T::n> calcForces(sys,*( (grav_t*) system_shared_data_pointer(this,compile_time_param) ) );

		// Local variables
		const int nbod = T::n;
		// Body number
		const int b = thread_body_idx(nbod);
		// Component number
		const int c = thread_component_idx(nbod);

		// local variables
		monitor_t montest(_mon_params,sys,*_log) ;


		// local information per component per body
		double pos = 0.0, vel = 0.0 , acc0 = 0.0, jerk0 = 0.0;
		if( (b < nbod) && (c < 3) )
			{ pos = sys[b][c].pos(); vel = sys[b][c].vel(); }


//		if( thread_in_system()==0  )  {
		    montest( thread_in_system() );
//		    }

		////////// INTEGRATION //////////////////////

		// Calculate acceleration and jerk
		calcForces(thread_in_system(),b,c,pos,vel,acc0,jerk0);

		for(int iter = 0 ; (iter < _max_iterations) && sys.is_active() ; iter ++ ) 
		{
			double h = _time_step;

			if( sys.time() + h > _destination_time ) {
				h = _destination_time - sys.time();
			}

			
			// Initial Evaluation
			///calcForces(thread_in_system(),b,c,pos,vel,acc0,jerk0);

			// Predict 
			pos = pos +  h*(vel+(h*0.5)*(acc0+(h/3.0)*jerk0));
			vel = vel +  h*(acc0+(h*0.5)*jerk0);

			double pre_pos = pos, pre_vel = vel;

			double acc1,jerk1;
			{
				// Evaluation
				calcForces(thread_in_system(),b,c,pos,vel,acc1,jerk1);
				
				// Correct
#if 1 // OLD
				pos = pre_pos + (0.1-0.25) * (acc0 - acc1) * h * h - 1.0/60.0 * ( 7.0 * jerk0 + 2.0 * jerk1 ) * h * h * h;
				vel = pre_vel + ( -0.5 ) * (acc0 - acc1 ) * h -  1.0/12.0 * ( 5.0 * jerk0 + jerk1 ) * h * h;
#else
				pos = pre_pos + ( (0.1-0.25) * (acc0 - acc1) - 1.0/60.0 * ( 7.0 * jerk0 + 2.0 * jerk1 ) * h) * h * h;
				vel = pre_vel + (( -0.5 ) * (acc0 - acc1 ) -  1.0/12.0 * ( 5.0 * jerk0 + jerk1 ) * h )* h ;
#endif
			}
			{
				// Evaluation
				calcForces(thread_in_system(),b,c,pos,vel,acc1,jerk1);
				
				// Correct
#if 1 // OLD
				pos = pre_pos + (0.1-0.25) * (acc0 - acc1) * h * h - 1.0/60.0 * ( 7.0 * jerk0 + 2.0 * jerk1 ) * h * h * h;
				vel = pre_vel + ( -0.5 ) * (acc0 - acc1 ) * h -  1.0/12.0 * ( 5.0 * jerk0 + jerk1 ) * h * h;
#else
				pos = pre_pos + ((0.1-0.25) * (acc0 - acc1) - 1.0/60.0 * ( 7.0 * jerk0 + 2.0 * jerk1 ) * h )* h * h ;
				vel = pre_vel + (( -0.5 ) * (acc0 - acc1 ) -  1.0/12.0 * ( 5.0 * jerk0 + jerk1 ) * h ) * h ;
#endif
			}
			acc0 = acc1, jerk0 = jerk1;

			// Finalize the step
			if( (b < nbod) && (c < 3) )
				{ sys[b][c].pos() = pos; sys[b][c].vel() = vel; }
			if( thread_in_system()==0 ) 
				sys.time() += h;
			__syncthreads();
			montest( thread_in_system() );  
			__syncthreads();
			if( sys.is_active() && thread_in_system()==0 )  {
			    if( sys.time() >= _destination_time ) 
			    {	sys.set_inactive(); }
			}

			__syncthreads();


		}

	}


};


typedef gpulog::device_log L;
using namespace monitors;

integrator_plugin_initializer<hermite< stop_on_ejection<L> > >
	hermite_plugin("hermite");

integrator_plugin_initializer<hermite< stop_on_ejection_or_close_encounter<L> > >
	hermite_close_encounter_plugin("hermite_close_encounter");

integrator_plugin_initializer<hermite< log_time_interval<L> > >
	hermite_log_plugin("hermite_log");


} } } // end namespace bppt :: integrators :: swarm
