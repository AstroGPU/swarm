#include "hip/hip_runtime.h"
/*************************************************************************
 * Copyright (C) 2011 by Saleh Dindar and the Swarm-NG Development Team  *
 *                                                                       *
 * This program is free software; you can redistribute it and/or modify  *
 * it under the terms of the GNU General Public License as published by  *
 * the Free Software Foundation; either version 3 of the License.        *
 *                                                                       *
 * This program is distributed in the hope that it will be useful,       *
 * but WITHOUT ANY WARRANTY; without even the implied warranty of        *
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the         *
 * GNU General Public License for more details.                          *
 *                                                                       *
 * You should have received a copy of the GNU General Public License     *
 * along with this program; if not, write to the                         *
 * Free Software Foundation, Inc.,                                       *
 * 59 Temple Place - Suite 330, Boston, MA  02111-1307, USA.             *
 ************************************************************************/

#include "hp.hpp"
#include "static_accjerk.hpp"
#include "swarmlog.h"


namespace swarm {
namespace hp {

struct FixedTimeStep {
	const static bool adaptive_time_step = false;
	const static bool conditional_accept_step = false;
};

struct AdaptiveTimeStep {
	const static bool adaptive_time_step = true;
	const static bool conditional_accept_step = true;
};

template < class AdaptationStyle >
class rkck: public integrator {
	typedef integrator base;
	private:
	double _min_time_step;
	double _max_time_step;
	double _error_tolerance;

	public:

	/*! Public constructor to configure the integrator
	 *  Read the configuration provided and set-up integrator variables
	 *
	 */
	rkck(const config& cfg): base(cfg),_min_time_step(0.001),_max_time_step(0.1) {
		if(!cfg.count("min time step")) ERROR("Integrator hp_rkck requires a min timestep ('min time step' keyword in the config file).");
		_min_time_step = atof(cfg.at("min time step").c_str());
		if(!cfg.count("max time step")) ERROR("Integrator hp_rkck requires a max timestep ('max time step' keyword in the config file).");
		_max_time_step = atof(cfg.at("max time step").c_str());

		if(!cfg.count("error tolerance")) ERROR("Integrator hp_rkck requires a error tolerance ('error tolerance' keyword in the config file).");
		_error_tolerance = atof(cfg.at("error tolerance").c_str());
	}


	virtual void launch_integrator() {
		launch_templatized_integrator(this);
	}

	/*! Integrator Kernel to be run on GPU
	 *  
	 *
	 */
	 template<class T >
	__device__ void kernel(T a)  {

		const int nbod = T::n;

////////////////////// RKCK Constants /////////////////////////////
	// Cash-Karp constants From GSL
	// Step 1 coefficient
	const double b1 = 1.0 / 5.0;
	// Step 2 coefficient
	const double b2[]  = { 3.0 / 40.0, 9.0 / 40.0 };
	// Step 3 coefficient
	const double b3[]  = { 0.3, -0.9, 1.2 };
	// Step 4 coefficient
	const double b4[]  = { -11.0 / 54.0, 2.5, -70.0 / 27.0, 35.0 / 27.0 };
	// Step 5 coefficient
	const double b5[]  = { 1631.0 / 55296.0, 175.0 / 512.0, 575.0 / 13824.0, 44275.0 / 110592.0, 253.0 / 4096.0 };
	// Step 6 coefficient
	const double b6[]  = { 37.0 / 378.0, 0, 250.0 / 621.0, 125.0 / 594.0, 0 , 512.0 / 1771.0 } ;
	// Error estimation coefficients
	const double ecc[] = { 37.0 / 378.0 - 2825.0 / 27648.0, 0.0, 250.0 / 621.0 - 18575.0 / 48384.0, 125.0 / 594.0 - 13525.0 / 55296.0, -277.00 / 14336.0, 512.0 / 1771.0 - 0.25 };


		/////////////// FETCH LOCAL VARIABLES ///////////////////

		int thr = thread_in_system();

		if(sysid() >= _gpu_ens->nsys()) { return; }
		ensemble::systemref sys ( (*_gpu_ens)[sysid()] );

		// Body/Component Grid
		// Body number
		int b = thr / 3 ;
		// Component number
		int c = thr % 3 ;
		bool body_component_grid = b < nbod;

		// i,j pairs Grid
		int ij = thr;

		// shared memory allocation
		extern __shared__ char shared_mem[];
		char*  system_shmem =( shared_mem + sysid_in_block() * shmem_per_system(nbod) );

		double (&shared_mag)[2][nbod][3] = * (double (*)[2][nbod][3]) system_shmem;

		double t_start = sys.time(), t = t_start;
		double t_end = min(t_start + _destination_time,sys.time_end());
		double time_step = _max_time_step;

		// local information per component per body
		double pos = 0, vel = 0;
		if( body_component_grid )
			pos = sys[b].p(c), vel = sys[b].v(c);


		////////// INTEGRATION //////////////////////

		// Calculate acceleration and jerk
		Gravitation<nbod> calcForces(sys,system_shmem);

		while(t < t_end){
			double h = min(time_step, t_end - t);



			//// RKCK   integrate system  ////////////////////////////////////////////////////////////////
			double p0 = pos, v0 = vel;

			// Step 1
			double k1_acc = calcForces.acc(ij,b,c,p0,v0);
			double k1_vel = v0;

			double p1 = pos + h * b1 * k1_vel;
			double v1 = vel + h * b1 * k1_acc;

			// Step 2
			double k2_acc = calcForces.acc(ij,b,c,p1,v1);
			double k2_vel = v1;

			double p2 = pos + h * ( b2[0] * k1_vel + b2[1] * k2_vel );
			double v2 = vel + h * ( b2[0] * k1_acc + b2[1] * k2_acc );

			// Step 3
			double k3_acc = calcForces.acc(ij,b,c,p2,v2);
			double k3_vel = v2;

			double p3 = pos + h * ( b3[0] * k1_vel + b3[1] * k2_vel + b3[2] * k3_vel );
			double v3 = vel + h * ( b3[0] * k1_acc + b3[1] * k2_acc + b3[2] * k3_acc );

			// Step 4
			double k4_acc = calcForces.acc(ij,b,c,p3,v3);
			double k4_vel = v3;

			double p4 = pos + h * ( b4[0] * k1_vel + b4[1] * k2_vel + b4[2] * k3_vel + b4[3] * k4_vel );
			double v4 = vel + h * ( b4[0] * k1_acc + b4[1] * k2_acc + b4[2] * k3_acc + b4[3] * k4_acc );

			// Step 5
			double k5_acc = calcForces.acc(ij,b,c,p4,v4);
			double k5_vel = v4;

			double p5 = pos + h * ( b5[0] * k1_vel + b5[1] * k2_vel + b5[2] * k3_vel + b5[3] * k4_vel + b5[4] * k5_vel );
			double v5 = vel + h * ( b5[0] * k1_acc + b5[1] * k2_acc + b5[2] * k3_acc + b5[3] * k4_acc + b5[4] * k5_acc );

			// Step 6
			double k6_acc = calcForces.acc(ij,b,c,p5,v5);
			double k6_vel = v5;

			double p6 = pos + h * ( b6[0] * k1_vel + b6[1] * k2_vel + b6[2] * k3_vel + b6[3] * k4_vel + b6[4] * k5_vel + b6[5] * k6_vel );
			double v6 = vel + h * ( b6[0] * k1_acc + b6[1] * k2_acc + b6[2] * k3_acc + b6[3] * k4_acc + b6[4] * k5_acc + b6[5] * k6_acc );


			// Error estimate
			double pos_error = h * ( ecc[0] * k1_vel + ecc[1] * k2_vel + ecc[2] * k3_vel + ecc[3] * k4_vel + ecc[4] * k5_vel + ecc[5] * k6_vel );
			double vel_error = h * ( ecc[0] * k1_acc + ecc[1] * k2_acc + ecc[2] * k3_acc + ecc[3] * k4_acc + ecc[4] * k5_acc + ecc[5] * k6_acc );


			bool accept_step = true;

			if( AdaptationStyle::adaptive_time_step ) {
				////////////////////////  Adapting Time step algorithm /////////////////////////////
				const int   integrator_order = 5;
				//! Value used as power in formula to produce larger time step
				const float step_grow_power = -1./(integrator_order+1.);
				//! Value used as power in formula to produce smaller time step
				const float step_shrink_power = -1./integrator_order;
				//! Safety factor to prevent extreme changes in time step
				const float step_guess_safety_factor = 0.9;
				//! Maximum growth of step size allowed at a time
				const float step_grow_max_factor = 5.0; 
				//! Maximum shrinkage of step size allowed at a time
				const float step_shrink_min_factor = 0.2; 

				//  Calculate the error estimate
				if( body_component_grid ) {

					sys[b].p(c) = p6 * p6 , sys[b].v(c) = v6 * v6;
					shared_mag[0][b][c] = pos_error * pos_error;
					shared_mag[1][b][c] = vel_error * vel_error;

					__syncthreads();
					if ( (c == 0) && (b == 0) ) {

						double max_error = 0;
						for(int i = 0; i < nbod ; i++){
							double pos_error_mag = shared_mag[0][i][0] + shared_mag[0][i][1] + shared_mag[0][i][2];
							double pos_mag = sys[i].p(0) + sys[i].p(1) + sys[i].p(2);
							double pe = pos_error_mag / pos_mag ;

							double vel_error_mag = shared_mag[1][i][0] + shared_mag[1][i][1] + shared_mag[1][i][2];
							double vel_mag = sys[i].v(0) + sys[i].v(1) + sys[i].v(2);
							double ve = vel_error_mag / vel_mag ;

							max_error = max ( max( pe, ve) , max_error );
						}

						double normalized_error = max_error / _error_tolerance;

						// Calculate New time_step
						double step_guess_power = (normalized_error<1.) ? step_grow_power : step_shrink_power;

						/// factor of 0.5 below due to use of squares in calculate_normalized_error, should we change to match gsl?
						/// gsl uses 1.1, but that seems dangerous, any reason we shouldn't use 1?
						double step_change_factor = ((normalized_error<0.5)||(normalized_error>1.0)) ? step_guess_safety_factor*pow(normalized_error,0.5*step_guess_power) : 1.0;


						//// Update the time step
						double new_time_step = (normalized_error>1.) ? max( time_step * max(step_change_factor,step_shrink_min_factor), _min_time_step ) 
							: min( time_step * max(min(step_change_factor,step_grow_max_factor),1.0), _max_time_step );

						bool accept = ( normalized_error < 1.0 ) || (abs(time_step - new_time_step) < 1e-10) ;

						shared_mag[0][0][0] = accept ? 0.0 : 1.0;
						shared_mag[0][0][1] = new_time_step;
					}

				}
				__syncthreads();

				time_step = shared_mag[0][0][1];
				accept_step = AdaptationStyle::conditional_accept_step ? (shared_mag[0][0][0] == 0.0) : true;
				////////////////////////// End of Adaptive time step algorithm  ////////////////////////////////////////////
			}


			if ( accept_step ) {

				// Set the new positions and velocities and time
				pos = p6;
				vel = v6;
				t += h;

				if( body_component_grid )
					sys[b].p(c) = pos, sys[b].v(c) = vel;

				if(thr == 0) 
					if(log::needs_output(*_gpu_ens, t, sysid()))
					{
						sys.set_time(t);
						log::output_system(*_gpu_log, *_gpu_ens, t, sysid());
					}
			}

		}

		if(thr == 0) 
			sys.set_time(t);
	}

};

/*!
 * \brief Factory to create double/single/mixed rkck gpu integrator based on precision
 *
 * @param[in] cfg configuration class
 *
 * @return        pointer to integrator cast to integrator*
 */
extern "C" integrator *create_hp_rkck_fixed(const config &cfg)
{
	return new rkck< FixedTimeStep> (cfg);
}

extern "C" integrator *create_hp_rkck_adaptive(const config &cfg)
{
	return new rkck< AdaptiveTimeStep> (cfg);
}

}
}

