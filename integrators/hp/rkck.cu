#include "hip/hip_runtime.h"
/*************************************************************************
 * Copyright (C) 2011 by Saleh Dindar and the Swarm-NG Development Team  *
 *                                                                       *
 * This program is free software; you can redistribute it and/or modify  *
 * it under the terms of the GNU General Public License as published by  *
 * the Free Software Foundation; either version 3 of the License.        *
 *                                                                       *
 * This program is distributed in the hope that it will be useful,       *
 * but WITHOUT ANY WARRANTY; without even the implied warranty of        *
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the         *
 * GNU General Public License for more details.                          *
 *                                                                       *
 * You should have received a copy of the GNU General Public License     *
 * along with this program; if not, write to the                         *
 * Free Software Foundation, Inc.,                                       *
 * 59 Temple Place - Suite 330, Boston, MA  02111-1307, USA.             *
 ************************************************************************/

#include "hp.hpp"
#include "static_accjerk.hpp"
#include "swarmlog.h"


namespace swarm {
namespace hp {


#define ADAPTIVE_TIMESTEP	


template<int i>
struct TT {
	const static int n = i;
};

template<class I,class T>
__global__ void generic_kernel(I* integ,T a) {
	integ->kernel(a);
}

class rkck: public integrator {
	private:
	double _min_time_step;
	double _max_time_step;
	double _error_tolerance;
	double _destination_time;

	public:
	 template<class T >
	__device__ void kernel(T a)  {

		const int nbod = T::n;

////////////////////// RKCK Constants /////////////////////////////
	// Cash-Karp constants From GSL
	// Step 1 coefficient
	const double b1 = 1.0 / 5.0;
	// Step 2 coefficient
	const double b2[]  = { 3.0 / 40.0, 9.0 / 40.0 };
	// Step 3 coefficient
	const double b3[]  = { 0.3, -0.9, 1.2 };
	// Step 4 coefficient
	const double b4[]  = { -11.0 / 54.0, 2.5, -70.0 / 27.0, 35.0 / 27.0 };
	// Step 5 coefficient
	const double b5[]  = { 1631.0 / 55296.0, 175.0 / 512.0, 575.0 / 13824.0, 44275.0 / 110592.0, 253.0 / 4096.0 };
	// Step 6 coefficient
	const double b6[]  = { 37.0 / 378.0, 0, 250.0 / 621.0, 125.0 / 594.0, 0 , 512.0 / 1771.0 } ;
	// Error estimation coefficients
	const double ecc[] = { 37.0 / 378.0 - 2825.0 / 27648.0, 0.0, 250.0 / 621.0 - 18575.0 / 48384.0, 125.0 / 594.0 - 13525.0 / 55296.0, -277.00 / 14336.0, 512.0 / 1771.0 - 0.25 };


		/////////////// FETCH LOCAL VARIABLES ///////////////////

		int thr = thread_in_system();

		if(sysid() >= _gpu_ens->nsys()) { return; }
		ensemble::systemref sys ( (*_gpu_ens)[sysid()] );

		// Body/Component Grid
		// Body number
		int b = thr / 3 ;
		// Component number
		int c = thr % 3 ;
		bool body_component_grid = b < nbod;

		// i,j pairs Grid
		int ij = thr;

		// shared memory allocation
		extern __shared__ char shared_mem[];
		char*  system_shmem =( shared_mem + sysid_in_block() * shmem_per_system(nbod) );

		double (&shared_mag)[nbod][3] = * (double (*)[nbod][3]) system_shmem;
		double (&shared_err)[nbod][2] = * (double (*)[nbod][2]) system_shmem;

		double t_start = sys.time(), t = t_start;
		double t_end = min(t_start + _destination_time,sys.time_end());
		double time_step = _min_time_step;

		// local information per component per body
		double pos = 0, vel = 0 , acc = 0, jerk = 0;
		if( body_component_grid )
			pos = sys[b].p(c), vel = sys[b].v(c);


		////////// INTEGRATION //////////////////////

		// Calculate acceleration and jerk
		Gravitation<nbod> calcForces(sys,system_shmem);
		calcForces(ij,b,c,pos,vel,acc,jerk);

		while(t < t_end){
			double h = min(time_step, t_end - t);



			//// RKCK   integrate system  ////////////////////////////////////////////////////////////////
			double p0 = pos, v0 = vel;

			// Step 1
			double k1_acc = calcForces.acc(ij,b,c,p0,v0);
			double k1_vel = v0;

			double p1 = pos + h * b1 * k1_vel;
			double v1 = vel + h * b1 * k1_acc;

			// Step 2
			double k2_acc = calcForces.acc(ij,b,c,p1,v1);
			double k2_vel = v1;

			double p2 = pos + h * ( b2[0] * k1_vel + b2[1] * k2_vel );
			double v2 = vel + h * ( b2[0] * k1_acc + b2[1] * k2_acc );

			// Step 3
			double k3_acc = calcForces.acc(ij,b,c,p2,v2);
			double k3_vel = v2;

			double p3 = pos + h * ( b3[0] * k1_vel + b3[1] * k2_vel + b3[2] * k3_vel );
			double v3 = vel + h * ( b3[0] * k1_acc + b3[1] * k2_acc + b3[2] * k3_acc );

			// Step 4
			double k4_acc = calcForces.acc(ij,b,c,p3,v3);
			double k4_vel = v3;

			double p4 = pos + h * ( b4[0] * k1_vel + b4[1] * k2_vel + b4[2] * k3_vel + b4[3] * k4_vel );
			double v4 = vel + h * ( b4[0] * k1_acc + b4[1] * k2_acc + b4[2] * k3_acc + b4[3] * k4_acc );

			// Step 5
			double k5_acc = calcForces.acc(ij,b,c,p4,v4);
			double k5_vel = v4;

			double p5 = pos + h * ( b5[0] * k1_vel + b5[1] * k2_vel + b5[2] * k3_vel + b5[3] * k4_vel + b5[4] * k5_vel );
			double v5 = vel + h * ( b5[0] * k1_acc + b5[1] * k2_acc + b5[2] * k3_acc + b5[3] * k4_acc + b5[4] * k5_acc );

			// Step 6
			double k6_acc = calcForces.acc(ij,b,c,p5,v5);
			double k6_vel = v5;

			double p6 = pos + h * ( b6[0] * k1_vel + b6[1] * k2_vel + b6[2] * k3_vel + b6[3] * k4_vel + b6[4] * k5_vel + b6[5] * k6_vel );
			double v6 = vel + h * ( b6[0] * k1_acc + b6[1] * k2_acc + b6[2] * k3_acc + b6[3] * k4_acc + b6[4] * k5_acc + b6[5] * k6_acc );


			// Error estimate
			double pos_error = h * ( ecc[0] * k1_vel + ecc[1] * k2_vel + ecc[2] * k3_vel + ecc[3] * k4_vel + ecc[4] * k5_vel + ecc[5] * k6_vel );
			double vel_error = h * ( ecc[0] * k1_acc + ecc[1] * k2_acc + ecc[2] * k3_acc + ecc[3] * k4_acc + ecc[4] * k5_acc + ecc[5] * k6_acc );


			////////////////////////  Adapting Time step algorithm /////////////////////////////
#ifdef ADAPTIVE_TIMESTEP
			const int   integrator_order = 5;
			//! Value used as power in formula to produce larger time step
			const float step_grow_power = -1./(integrator_order+1.);
			//! Value used as power in formula to produce smaller time step
			const float step_shrink_power = -1./integrator_order;
			//! Safety factor to prevent extreme changes in time step
			const float step_guess_safety_factor = 0.9;
			//! Maximum growth of step size allowed at a time
			const float step_grow_max_factor = 5.0; 
			//! Maximum shrinkage of step size allowed at a time
			const float step_shrink_min_factor = 0.2; 

			//  Calculate the error estimate
			if( body_component_grid ) {
				double pos_mag = 1, pos_error_mag = 0 , vel_mag = 1, vel_error_mag = 0;

				//// Compute pos magnitute in c=0 thread
				shared_mag[b][c] = p6 * p6;
				__syncthreads();
				if( c == 0 ) pos_mag = shared_mag[b][0] + shared_mag[b][1] + shared_mag[b][2];

				__syncthreads();

				//// Compute pos error magnitute in c=0 thread
				shared_mag[b][c] = pos_error * pos_error;
				__syncthreads();
				if( c == 0 ) pos_error_mag = shared_mag[b][0] + shared_mag[b][1] + shared_mag[b][2];

				__syncthreads();

				//// Compute vel magnitute in c=0 thread
				shared_mag[b][c] = v6 * v6;
				__syncthreads();
				if( c == 0 ) vel_mag = shared_mag[b][0] + shared_mag[b][1] + shared_mag[b][2];

				__syncthreads();

				//// Compute vel error magnitute in c=0 thread
				shared_mag[b][c] = vel_error * vel_error;
				__syncthreads();
				if( c == 0 ) vel_error_mag = shared_mag[b][0] + shared_mag[b][1] + shared_mag[b][2];

				__syncthreads();

				//// Write error estimates for each body to shared memory
				if( c == 0 ) shared_err[b][0] = pos_error_mag / pos_mag ;
				if( c == 0 ) shared_err[b][1] = vel_error_mag / vel_mag ;

				__syncthreads();

				// Calculate the maximum error in one of the threads
				if( b == 0 && c == 0 )  {
					double max_error = 0;
					for(int i = 0; i < nbod; i++)
						for(int j = 0; j < 2; j++)
							if(shared_err[i][j] > max_error) max_error = shared_err[i][j];

					double normalized_error = max_error / _error_tolerance;

					// Calculate New time_step
					double step_guess_power = (normalized_error<1.) ? step_grow_power : step_shrink_power;

					/// factor of 0.5 below due to use of squares in calculate_normalized_error, should we change to match gsl?
					/// gsl uses 1.1, but that seems dangerous, any reason we shouldn't use 1?
					double step_change_factor = ((normalized_error<0.5)||(normalized_error>1.0)) ? step_guess_safety_factor*pow(normalized_error,0.5*step_guess_power) : 1.0;


					//// Update the time step
					double new_time_step = (normalized_error>1.) ? max( time_step * max(step_change_factor,step_shrink_min_factor), _min_time_step ) 
						: min( time_step * max(min(step_change_factor,step_grow_max_factor),1.0), _max_time_step );

					bool accept_step = ( normalized_error < 1.0 ) || abs(time_step - new_time_step) < 1e-10;

					shared_err[0][0] = accept_step ? 0.0 : 1.0;
					shared_err[0][1] = new_time_step;
				}

				__syncthreads();
			}

			time_step = shared_err[0][1];
			bool accept_step = shared_err[0][0] == 0.0;
#else
			bool accept_step = true;
#endif
			////////////////////////// End of Adaptive time step algorithm  ////////////////////////////////////////////


			if ( accept_step ) {

				// Set the new positions and velocities and time
				pos = p6;
				vel = v6;
				t += h;

				if( body_component_grid )
					sys[b].p(c) = pos, sys[b].v(c) = vel;

				if(thr == 0) 
					if(log::needs_output(*_gpu_ens, t, sysid()))
					{
						sys.set_time(t);
						log::output_system(*_gpu_log, *_gpu_ens, t, sysid());
					}
			}

		}

		if(thr == 0) 
			sys.set_time(t);
	}
	rkck(const config& cfg): integrator(cfg),_min_time_step(0.001),_max_time_step(0.1) {
		if(!cfg.count("min time step")) ERROR("Integrator hp_rkck requires a min timestep ('min time step' keyword in the config file).");
		_min_time_step = atof(cfg.at("min time step").c_str());
		if(!cfg.count("max time step")) ERROR("Integrator hp_rkck requires a max timestep ('max time step' keyword in the config file).");
		_max_time_step = atof(cfg.at("max time step").c_str());

		if(!cfg.count("error tolerance")) ERROR("Integrator hp_rkck requires a error tolerance ('error tolerance' keyword in the config file).");
		_error_tolerance = atof(cfg.at("error tolerance").c_str());
	}


	template<class T>
	void launch_template(T a,rkck* gpu_integ)
	{


		if(_ens->nbod() == T::n) 
			generic_kernel<<<gridDim(), threadDim(), shmemSize() >>>(gpu_integ,a);

	}

	virtual void launch_integrator(const double& destination_time){
			// flush CPU/GPU output logs
			log::flush(log::memory | log::if_full);

			_destination_time = destination_time ;

			if(_ens->nbod() <= 3){
				rkck* integ;
				hipMalloc(&integ,sizeof(rkck));
				hipMemcpy(integ,this,sizeof(rkck),hipMemcpyHostToDevice);
				launch_template(TT<3>(),integ);
				hipFree(integ);
			} else {
				// How do we get an error message out of here?
				ERROR("Invalid number of bodies. (only up to 10 bodies per system)");
			}

			// flush CPU/GPU output logs
			log::flush(log::memory);
	}




};

/*!
 * \brief Factory to create double/single/mixed rkck gpu integrator based on precision
 *
 * @param[in] cfg configuration class
 *
 * @return        pointer to integrator cast to integrator*
 */
extern "C" integrator *create_hp_rkck(const config &cfg)
{
	return new rkck(cfg);
}

}
}

