#include "hip/hip_runtime.h"
/*************************************************************************
 * Copyright (C) 2011 by Saleh Dindar and the Swarm-NG Development Team  *
 *                                                                       *
 * This program is free software; you can redistribute it and/or modify  *
 * it under the terms of the GNU General Public License as published by  *
 * the Free Software Foundation; either version 3 of the License.        *
 *                                                                       *
 * This program is distributed in the hope that it will be useful,       *
 * but WITHOUT ANY WARRANTY; without even the implied warranty of        *
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the         *
 * GNU General Public License for more details.                          *
 *                                                                       *
 * You should have received a copy of the GNU General Public License     *
 * along with this program; if not, write to the                         *
 * Free Software Foundation, Inc.,                                       *
 * 59 Temple Place - Suite 330, Boston, MA  02111-1307, USA.             *
 ************************************************************************/

#include "hp/bppt.hpp"
#include "hp/helpers.hpp"
#include "hp/gravitation.hpp"
#include "stop_on_ejection.hpp"


namespace swarm {
namespace hp {

namespace gpu {
namespace bppt {

template< class _Stopper >
class hermite: public integrator {
	typedef integrator base;
	typedef  _Stopper stopper_t;
	private:
	double _time_step;
	int _iteration_count;
	stopper_t _stopper;

	public:
	hermite(const config& cfg): base(cfg),_time_step(0.001), _stopper(cfg) {
		if(!cfg.count("time step")) ERROR("Integrator gpu_hermite requires a timestep ('time step' keyword in the config file).");
		_time_step = atof(cfg.at("time step").c_str());
	}

	virtual void launch_integrator() {
		_iteration_count = _destination_time / _time_step;
		launch_templatized_integrator(this);
	}


	template<class T>
	__device__ void kernel(T a){
		// References to Ensemble and Shared Memory
		ensemble::SystemRef sys = _dens[sysid()];
		typedef typename Gravitation<T::n>::shared_data grav_t;
		Gravitation<T::n> calcForces(sys,*( (grav_t*) system_shared_data_pointer(a) ) );

		// Local variables
		const int nbod = T::n;
		// Body number
		int b = thread_body_idx(nbod);
		// Component number
		int c = thread_component_idx(nbod);
		int ij = thread_in_system();
		bool body_component_grid = (b < nbod) && (c < 3);
		bool first_thread_in_system = thread_in_system() == 0;


		// local variables
		typename stopper_t::tester stopper_tester = _stopper.get_tester(sys) ;

		// local information per component per body
		double pos = 0, vel = 0 , acc0 = 0, jerk0 = 0;
		if( body_component_grid )
			pos = sys[b][c].pos() , vel = sys[b][c].vel();


		////////// INTEGRATION //////////////////////

		// Calculate acceleration and jerk
		calcForces(ij,b,c,pos,vel,acc0,jerk0);

		for(int iter = 0 ; (iter < _iteration_count) && sys.active() ; iter ++ ) {
			double h = _time_step;
			// can't use this one because t might go past t_end
			// double h = min(_time_step, t_end - t);

			
			// Initial Evaluation
			///calcForces(ij,b,c,pos,vel,acc0,jerk0);

			// Predict 
			pos = pos +  h*(vel+(h*0.5)*(acc0+(h/3.)*jerk0));
			vel = vel +  h*(acc0+(h*0.5)*jerk0);

			double pre_pos = pos, pre_vel = vel;

			double acc1,jerk1;
			{
				// Evaluation
				calcForces(ij,b,c,pos,vel,acc1,jerk1);

				// Correct
				pos = pre_pos + (.1-.25) * (acc0 - acc1) * h * h - 1/60.0 * ( 7 * jerk0 + 2 * jerk1 ) * h * h * h;
				vel = pre_vel + ( -.5 ) * (acc0 - acc1 ) * h -  1/12.0 * ( 5 * jerk0 + jerk1 ) * h * h;
			}
			{
				// Evaluation
				calcForces(ij,b,c,pos,vel,acc1,jerk1);

				// Correct
				pos = pre_pos + (.1-.25) * (acc0 - acc1) * h * h - 1/60.0 * ( 7 * jerk0 + 2 * jerk1 ) * h * h * h;
				vel = pre_vel + ( -.5 ) * (acc0 - acc1 ) * h -  1/12.0 * ( 5 * jerk0 + jerk1 ) * h * h;
			}
			acc0 = acc1, jerk0 = jerk1;

			// Finalize the step
			if( body_component_grid )
				sys[b][c].pos() = pos , sys[b][c].vel() = vel;
			if( first_thread_in_system ) 
				sys.time() += h;

			if( first_thread_in_system ) 
				sys.active() = ! stopper_tester() ;

			__syncthreads();


		}

	}


};

/*!
 * \brief Factory to create double/single/mixed hermite gpu integrator based on precision
 *
 * @param[in] cfg configuration class
 *
 * @return        pointer to integrator cast to integrator*
 */
extern "C" integrator *create_hp_hermite(const config &cfg)
{
	return new hermite< stop_on_ejection >(cfg);
}

}
}
}
}
