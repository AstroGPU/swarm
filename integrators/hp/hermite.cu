#include "hip/hip_runtime.h"
/*************************************************************************
 * Copyright (C) 2011 by Saleh Dindar and the Swarm-NG Development Team  *
 *                                                                       *
 * This program is free software; you can redistribute it and/or modify  *
 * it under the terms of the GNU General Public License as published by  *
 * the Free Software Foundation; either version 3 of the License.        *
 *                                                                       *
 * This program is distributed in the hope that it will be useful,       *
 * but WITHOUT ANY WARRANTY; without even the implied warranty of        *
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the         *
 * GNU General Public License for more details.                          *
 *                                                                       *
 * You should have received a copy of the GNU General Public License     *
 * along with this program; if not, write to the                         *
 * Free Software Foundation, Inc.,                                       *
 * 59 Temple Place - Suite 330, Boston, MA  02111-1307, USA.             *
 ************************************************************************/

#include "hp.hpp"
#include "static_accjerk.hpp"
#include "swarmlog.h"


namespace swarm {
namespace hp {

template<int nbod>
__global__ static void hermite_kernel(ensemble* ens,gpulog::device_log* dlog,double destination_time, double time_step){

	/////////////// FETCH LOCAL VARIABLES ///////////////////

	int thr = thread_in_system();

	if(sysid() >= ens->nsys()) { return; }
	ensemble::systemref sys ( (*ens)[sysid()] );

	// Body/Component Grid
	// Body number
	int b = thr / 3 ;
	// Component number
	int c = thr % 3 ;
	bool body_component_grid = b < nbod;

	// i,j pairs Grid
	int ij = thr;

	// shared memory allocation
	extern __shared__ char shared_mem[];

	double t_start = sys.time(), t = t_start;
	double t_end = min(t_start + destination_time,sys.time_end());

	// local information per component per body
	double pos = 0, vel = 0 , acc = 0, jerk = 0;
	if( body_component_grid )
		pos = sys[b].p(c), vel = sys[b].v(c);


	////////// INTEGRATION //////////////////////

	// Calculate acceleration and jerk
	Gravitation<nbod> calcForces(sys,shared_mem);
	calcForces(ij,b,c,pos,vel,acc,jerk);

	while(t < t_end){
		for(int k = 0; k < 2; k++)
		{
			double h = min(time_step, t_end - t);
			double pos_old = pos, vel_old = vel, acc_old = acc,jerk_old = jerk;

			// Predict 
			pos = pos_old +  h*(vel_old+(h*0.5)*(acc+(h/3.)*jerk));
			vel = vel_old +  h*(acc+(h*0.5)*jerk);

			// Do evaluation and correction two times (PEC2)
			for(int l = 0; l < 2; l++)
			{

				// Calculate acceleration and jerk using shared memory
				calcForces(ij,b,c,pos,vel,acc,jerk);

				// Correct
				pos = pos_old + (h*0.5) * ( (vel_old + vel) 
						+ (h*7.0/30.)*( (acc_old-acc) + (h/7.) * (jerk_old+jerk)));
				vel = vel_old + (h*0.5) * ( (acc_old+acc) + (h/6.) * (jerk_old-jerk));

			}
			t += h;
		}

		if( body_component_grid )
			sys[b].p(c) = pos, sys[b].v(c) = vel;

		if(thr == 0) 
			if(log::needs_output(*ens, t, sysid()))
			{
				sys.set_time(t);
				log::output_system(*dlog, *ens, t, sysid());
			}

	}

	if(thr == 0) 
		sys.set_time(t);
}

class hermite: public integrator {
	private:
	double _time_step;

	public:
	hermite(const config& cfg): integrator(cfg),_time_step(0.001) {
		if(!cfg.count("time step")) ERROR("Integrator gpu_hermite requires a timestep ('time step' keyword in the config file).");
		_time_step = atof(cfg.at("time step").c_str());
	}

	template<int nbod>
	void launch_template(const double& destination_time)
	{
		if(_ens->nbod() == nbod) 
			hermite_kernel<nbod><<<gridDim(), threadDim(), shmemSize() >>>(_gpu_ens,_gpu_log,destination_time, _time_step);

	}

	virtual void launch_integrator(const double& destination_time){
			// flush CPU/GPU output logs
			log::flush(log::memory | log::if_full);

			if(_ens->nbod() <= 3){
				launch_template<3>(destination_time);
			} else {
				// How do we get an error message out of here?
				ERROR("Invalid number of bodies. (only up to 10 bodies per system)");
			}

			// flush CPU/GPU output logs
			log::flush(log::memory);
	}
};

/*!
 * \brief Factory to create double/single/mixed hermite gpu integrator based on precision
 *
 * @param[in] cfg configuration class
 *
 * @return        pointer to integrator cast to integrator*
 */
extern "C" integrator *create_hp_hermite(const config &cfg)
{
	return new hermite(cfg);
}

}
}

