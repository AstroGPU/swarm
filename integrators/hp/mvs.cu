#include "hip/hip_runtime.h"
/*************************************************************************
 * Copyright (C) 2011 by Eric B. Ford and the Swarm-NG Development Team  *
 *                                                                       *
 * This program is free software; you can redistribute it and/or modify  *
 * it under the terms of the GNU General Public License as published by  *
 * the Free Software Foundation; either version 3 of the License.        *
 *                                                                       *
 * This program is distributed in the hope that it will be useful,       *
 * but WITHOUT ANY WARRANTY; without even the implied warranty of        *
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the         *
 * GNU General Public License for more details.                          *
 *                                                                       *
 * You should have received a copy of the GNU General Public License     *
 * along with this program; if not, write to the                         *
 * Free Software Foundation, Inc.,                                       *
 * 59 Temple Place - Suite 330, Boston, MA  02111-1307, USA.             *
 ************************************************************************/

#include "hp.hpp"
#include "static_accjerk.hpp"
#include "swarmlog.h"

// #define N_LAG 5.0 // integer n, for recommended Laguerre method
#define SIGN(a) ((a) < 0 ? -1 : 1)


namespace swarm {
namespace hp {

class mvs: public integrator {
	typedef integrator base;
	private:
	double _time_step;
	static const int _N_LAG = 5.0;


	public:
	mvs(const config& cfg): base(cfg),_time_step(0.001) {
		if(!cfg.count("time step")) ERROR("Integrator gpu_mvs requires a timestep ('time step' keyword in the config file).");
		_time_step = atof(cfg.at("time step").c_str());
	}

	virtual void launch_integrator() {
		launch_templatized_integrator(this);
	}



////////////////////////////////////////////////////////////////
// solving differential Kepler's equation
// in universal variable x
// using Laguerre method as outlined by Prusing+C eqn 2.43
// code adapted from Alice Quillen's Qymsym code 
// see http://astro.pas.rochester.edu/~aquillen/qymsym/
////////////////////////////////////////////////////////////////
#define MINR 1.0e-5 // minimum radius
#define MINDENOM 1e-8  // mininum denominator

__device__ double solvex(double r0dotv0, double alpha,
                double sqrtM1, double r0, double dt)
{
//   double smu = sqrt(M1);
   double smu = sqrtM1;
   double foo = 1.0 - r0*alpha;
   double sig0 = r0dotv0/smu;
   double x = sqrtM1*sqrtM1*dt*dt/r0; // initial guess could be improved 
// better initial guess depends on rperi which would have to be passed

   double u=1.0;
   // TODO: Could allow to exit loop early
   for(int i=0;i<7;i++){  // 7 iterations is probably overkill
			// as it always converges faster than this
     double x2,x3,alx2,Cp,Sp,F,dF,ddF,z;
     x2 = x*x;
     x3 = x2*x;
     alx2 = alpha*x2;
     Cp = C_prussing(alx2);
     Sp = S_prussing(alx2);
//   TODO: Test if this is faster/accurate     
//   SC_prussing(alx2,Sp,Cp);
     F = sig0*x2*Cp + foo*x3*Sp + r0*x - smu*dt; // eqn 2.41 PC
     dF = sig0*x*(1.0 - alx2*Sp)  + foo*x2*Cp + r0; // eqn 2.42 PC
     ddF = sig0*(1.0-alx2*Cp) + foo*x*(1.0 - alx2*Sp);
     z = fabs((_N_LAG - 1.0)*((_N_LAG - 1.0)*dF*dF - _N_LAG*F*ddF));
     z = sqrt(z);
     double denom = (dF + SIGN(dF)*z); 
     if (denom ==0.0) denom = MINDENOM;
     u = _N_LAG*F/denom; // equation 2.43 PC
     x -= u;
   }
//   if (isnan(x)) printf("solvex: is nan\n");
   return x;
}

// functions needed for kepstep
// code adapted from Alice Quillen's Qymsym code 
// see http://astro.pas.rochester.edu/~aquillen/qymsym/
// TODO: could merge into one function that uses sincos
__device__ double C_prussing(double y) // equation 2.40a Prussing + Conway
{
  if (fabs(y)<1e-4) return 1.0/2.0*(1.0 - y/12.0*(1.0 - y/30.0*(1.0 - y/56.0)));
  double u = sqrt(fabs(y));
  if (y>0.0) return (1.0- cos(u))/ y;
  else       return (cosh(u)-1.0)/-y;
}

__device__ double S_prussing(double y) // equation 2.40b Prussing +Conway
{
  if (fabs(y)<1e-4) return 1.0/6.0*(1.0 - y/20.0*(1.0 - y/42.0*(1.0 - y/72.0)));
  double u = sqrt(fabs(y));
  double u3 = u*u*u;
  if (y>0.0) return (u -  sin(u))/u3;
  else       return (sinh(u) - u)/u3;
}

__device__ void SC_prussing(double y, double& S, double &C) // equation 2.40a Prussing + Conway
{
  if (fabs(y)<1e-4) 
  {
     S = 1.0/6.0*(1.0 - y/20.0*(1.0 - y/42.0*(1.0 - y/72.0)));
     C = 1.0/2.0*(1.0 - y/12.0*(1.0 - y/30.0*(1.0 - y/56.0)));
     return;
  }
  double u = sqrt(fabs(y));
  double u3 = u*u*u;
  if (y>0.0) 
     {
     sincos(u,&S,&C);  // TODO: Need to verify called correctly
     S = (u -  S)/u3;
     C = (1.0- C)/ y;
     }
  else
     {
       S = (sinh(u) - u)/u3;
       C = (cosh(u)-1.0)/-y;
     }
  return;
}



///////////////////////////////////////////////////////////////
// advance a particle using f,g functions and universal variables
// for differental kepler's equation
//  has an error catch for r0=0 so can be run with central body
// Based on equations by Prussing, J. E. and Conway, B. A. 
// Orbital Mechanics 1993, Oxford University Press, NY,NY  chap 2 
// npos,nvel are new positions and velocity
// pos, vel are old ones
// code adapted from Alice Quillen's Qymsym code 
// see http://astro.pas.rochester.edu/~aquillen/qymsym/
///////////////////////////////////////////////////////////////
//__device__ void kepstep(double4 pos, double4 vel, double4* npos, double4* nvel, double deltaTime, double GM)
__device__ void drift_kepler(double& x_old, double& y_old, double& z_old, double& vx_old, double& vy_old, double& vz_old, const double sqrtGM, const double deltaTime)
{
   double x = x_old, y = y_old, z = z_old, vx = vx_old, vy = vy_old, vz = vz_old;
   // WARNING: Using softened potential
   double r0 = sqrt(x*x + y*y + z*z + MINR*MINR); // current radius
   double v2 = (vx*vx + vy*vy + vz*vz);  // current velocity
   double r0dotv0 = (x*vx + y*vy + z*vz);
   double GM = sqrtGM*sqrtGM;
   double alpha = (2.0/r0 - v2/GM);  // inverse of semi-major eqn 2.134 MD
// here alpha=1/a and can be negative
   double x_p = solvex(r0dotv0, alpha, sqrtGM, r0, deltaTime); // solve universal kepler eqn

//   double smu = sqrt(GM);  // before we cached sqrt(GM)
   double smu = sqrtGM; 
   double foo = 1.0 - r0*alpha;
   double sig0 = r0dotv0/smu;
   double x2 = x_p*x_p;
   double x3 = x2*x_p;
   double alx2 = alpha*x2;
   double Cp = C_prussing(alx2);
   double Sp = S_prussing(alx2);
//   TODO: Test if this is faster/accurate     
//   double Cp, Sp;
//   SC_prussing(alx2,Sp,Cp);
   double r = sig0*x_p*(1.0 - alx2*Sp)  + foo*x2*Cp + r0; // eqn 2.42  PC
   if (r < MINR) r=MINR;
// if dt == 0 then f=dgdt=1 and g=dfdt=0
// f,g functions equation 2.38a  PC
   double f_p= 1.0 - (x2/r0)*Cp;
   double g_p= deltaTime - (x3/smu)*Sp;
// dfdt,dgdt function equation 2.38b PC
   double dfdt;
   double dgdt = 1.0 - (x2/r)*Cp;
   if (fabs(g_p) > MINDENOM)
      // conservation of angular momentum means that f dfdt - g dfdt =1
      dfdt = (f_p*dgdt - 1.0)/g_p;
   else
      // dfdt,dgdt function equation 2.38b PC
      dfdt = x_p*smu/(r*r0)*(alx2*Sp - 1.0);
  
   x = f_p*x_old + g_p*vx_old;     // eqn 2.65 M+D
   y = f_p*y_old + g_p*vy_old;
   z = f_p*z_old + g_p*vz_old; 
   vx = dfdt*x_old + dgdt*vx_old;  // eqn 2.70 M+D
   vy = dfdt*y_old + dgdt*vy_old;
   vz = dfdt*z_old + dgdt*vz_old;

   // Replace values 
    x_old =  x;  y_old =  y;  z_old =  z;
   vx_old = vx; vy_old = vy; vz_old = vz;
}

	/*! Integrator Kernel to be run on GPU
	 *  
	 * TODO: Need to deal with energy conservation if input not in COM frame
	 */
	 template<class T >
	__device__ void kernel(T a)  {

		const int nbod = T::n;
		const bool allow_rewind = false;

		/////////////// FETCH LOCAL VARIABLES ///////////////////

		int thr = thread_in_system();

		/* WARNING: This is fine for now, but I wonder if this should be removed in the future.
		   I think it's here to prevent readingn outside the bounds of _gpu_ens.
		   But I worry that if we're usingn the threads in different ways, are we sure we'd never want to use this thread for something?  */
		if(sysid() >= _gpu_ens->nsys()) { return; }

		ensemble::systemref sys ( (*_gpu_ens)[sysid()] );

		// Body/Component Grid
		// Body number
		int b = thr / 3 ;  // index for parts w/ 1 thread per body
		int bb = b+1;      // index for parts w/ 1 thread per body excluding sun/central body
		// Component number
		int c = thr % 3 ;  
		bool body_component_grid = b < nbod;          // if needed for parts w/ 1 thread per body per component including sun/central body
		bool body_component_grid_no_sun = bb < nbod;  // if needed for parts w/ 1 thread per body per component excluding sun/central body
//		bool body_grid = thr < nbod;                  // if needed for parts w/ 1 thread per body including sun/central body

		// i,j pairs Grid
		// TODO: Be more clever about calculating accelerations
		//       Either avoid pairs with sun or specialize Gravitation?
		int ij = thr;      // index for parts w/ 1 thread per body pair


		// shared memory allocation
		extern __shared__ char shared_mem[];
		char*  system_shmem =( shared_mem + sysid_in_block() * integrator::shmem_per_system(nbod) );

		double t_start = sys.time(), t = t_start;
		double t_end = min(t_start + _destination_time,sys.time_end());

		// local information per component per body
		double pos_old, vel_old, acc_old, jerk_old; // needed if allowing rewindss
		double acc = 0., jerk = 0.;
		double sqrtGM = sqrt(sys[0].mass()); // TODO: Could parallelize. Worth it?

		// Shift into funky coordinate system (see A. Quillen's qymsym's tobary)
		if( (b==0) || body_component_grid_no_sun )
		   {
		   double sump = 0., sumv = 0., mtot = 0.;
		   for(int j=0;j<nbod;++j)   // TODO: Could parallelize. Worth it?
		      {
		      const double mj = sys[j].mass();
		      mtot += mj;
		      sump += mj*sys[j].p(c);
		      sumv += mj*sys[j].v(c);
		      }
		   if(b==0) // For sun only
		      {
		      sys[b].v(c) = sumv/mtot;
		      sys[b].p(c) = sump/mtot;
		      }
		   if( body_component_grid_no_sun ) // For all bodies except sun
		      {
		      sys[bb].v(c) -= sumv/mtot;
		      sys[bb].p(c) -= sys[0].p(c);
  		      }
		   }
		   __syncthreads();		

		////////// INTEGRATION //////////////////////
		// Calculate acceleration and jerk
		Gravitation<nbod> calcForces(sys,system_shmem);
                // precompute acc and jerk before enter loop, since will cache acc and jerk across loop itterations
		calcForces.calc_accel_no_sun(ij,bb,c,acc,jerk);

		unsigned int iter=0;  // Make sure don't get stuck in infinite loop
		while(t < t_end)      // Only enter loop if need to integrate
		{
		   double hby2 = 0.5*min(_time_step, t_end - t);
		   if(allow_rewind)   // Could be useful if later reject step
		     { 
		     if( body_component_grid )
			{ 
			pos_old = sys[b].p(c); vel_old = sys[b].v(c); 
			acc_old = acc;  jerk_old = jerk;
			}
		     }

		   // Drift Step (center-of-mass motion)
		   if( body_component_grid_no_sun )
		      {
		      double mv = 0.;
		      // TODO: In principle could parellalize.  Worth it?
		      for(int j=1;j<nbod;++j)
		      	 mv += sys[j].mass()*sys[j].v(c);
		      sys[bb].p(c) += mv*hby2/sys[0].mass();
		      }
		   __syncthreads();

		   // Kick Step (planet-planet interactions)
		   {
		   // TODO: Test that this call can be removed
		   // WARNING: If make changes, check that it's ok to not recompute
		   calcForces.calc_accel_no_sun(ij,bb,c,acc,jerk);
		   if( body_component_grid_no_sun )
		      {
		      sys[bb].v(c) +=  hby2*(acc+hby2*0.5*jerk);
		      }
		   }
		   __syncthreads();
  
		   // Kepler Drift Step (Keplerian orbit about sun/central body)
		   int bbb = thr+1;
		   if(bbb < nbod)  // Central body does not do Kepler drift
		   {
		      drift_kepler(sys[bbb].p(0),sys[bbb].p(1),sys[bbb].p(2),sys[bbb].v(0),sys[bbb].v(1),sys[bbb].v(2),sqrtGM,2.0*hby2);
		   }
		   __syncthreads();	   		   
		   
		   /* TODO: Eventually check for close encounters and 
		            if necessary undo, perform direct n-body, merge and resume
	  	            Or maybe only in separate integrator? */
		   bool need_to_rewind = false;
		   if( allow_rewind && need_to_rewind )
		     {
			sys[b].p(c) = pos_old; sys[b].v(c) = vel_old; 
			acc = acc_old;  jerk = jerk_old;
		     	++iter;
		   	if(iter>=_max_itterations_per_kernel_call) break;
			continue;
                     }

		   // Kick Step (planet-planet interactions)
		   {
		   calcForces.calc_accel_no_sun(ij,bb,c,acc,jerk);
		   if( body_component_grid_no_sun )
		      {
		      sys[bb].v(c) +=  hby2*(acc+hby2*0.5*jerk);
		      }
		   }
		   __syncthreads();

		   // Drift Step (center-of-mass motion)
		   if( body_component_grid_no_sun )
		      {
		      double mv = 0.;
		      // TODO: In principle could parellalize. Worth it?
		      for(int j=1;j<nbod;++j)
		      	 mv += sys[j].mass()*sys[j].v(c);
		      sys[bb].p(c) += mv*hby2/sys[0].mass();
		      }
		   __syncthreads();

		   // WARNING: Need to think about correct order of time updates, if add time dependnt forces
		   t += 2.*hby2;

		   ++iter;
		   if( log::needs_output(*_gpu_ens, t, sysid()) )
		      {
		      // Save working coordinates
		      double pos_tmp, vel_tmp;
		      if(body_component_grid )
			{ pos_tmp = sys[b].p(c); vel_tmp = sys[b].v(c); }

		      // Shift back from funky coordinate system (see A. Quillen's qymsym's tobary)
		      if( (b==0) || body_component_grid_no_sun )
		         {
		   	 const double m0 = sys[0].mass();
		   	 double sump = 0., sumv = 0., mtot = m0;
		   	 for(int j=1;j<nbod;++j)   // TODO: Could parallelize;  Worth it?
		      	    {
		      	    const double mj = sys[j].mass();
		      	    mtot += mj;
		      	    sump += mj*sys[j].p(c);
		      	    sumv += mj*sys[j].v(c);
		      	    }
		   	 if(b==0) // For sun only
		      	    {
		      	    sys[b].p(c) -= sump/mtot;
		      	    sys[b].v(c) -= sumv/m0;
		      	    }
		   	 if( body_component_grid_no_sun ) // For all bodies except sun
		      	    {
		      	    sys[bb].p(c) += sys[0].p(c) - sump/mtot;
		      	    sys[bb].v(c) += sys[0].v(c);
  		      	    }
		   	 }
		      __syncthreads();		      
		      if(thr == 0)
		         {
		         sys.set_time(t);
		         log::output_system(*_gpu_log, *_gpu_ens, t, sysid());
		         }
		      __syncthreads();
		      // Restore working coordinates
		      if(body_component_grid )
			{ sys[b].p(c) = pos_tmp; sys[b].v(c) = vel_tmp; }
		      __syncthreads();
		      }

		   if(iter>=_max_itterations_per_kernel_call) break;
		}

		// Shift back from funky coordinate system (see A. Quillen's qymsym's tobary)
		if( (b==0) || body_component_grid_no_sun )
		   {
		   const double m0 = sys[0].mass();
		   double sump = 0., sumv = 0., mtot = m0;
		   for(int j=1;j<nbod;++j)   // TODO: Could parallelize. Worth it?
		      {
		      const double mj = sys[j].mass();
		      mtot += mj;
		      sump += mj*sys[j].p(c);
		      sumv += mj*sys[j].v(c);
		      }
		   if(b==0) // For sun only
		      {
		      sys[b].p(c) -= sump/mtot;
		      sys[b].v(c) -= sumv/m0;
		      }
		   if( body_component_grid_no_sun ) // For all bodies except sun
		      {
		      sys[bb].p(c) += sys[0].p(c) - sump/mtot;
		      sys[bb].v(c) += sys[0].v(c);
  		      }
		   }
		   __syncthreads();

		if(thr == 0) 
		   sys.set_time(t);

	}

};

/*!
 * \brief Factory to create mvs gpu integrator from config class
 *
 * @param[in] cfg configuration class
 *
 * @return        pointer to integrator cast to integrator*
 */
/*
extern "C" integrator *create_hp_mvs_fixed(const config &cfg)
{
	return new mvs< FixedTimeStep> (cfg);
}

extern "C" integrator *create_hp_mvs_adaptive(const config &cfg)
{
	return new mvs< AdaptiveTimeStep> (cfg);
}
*/
extern "C" integrator *create_hp_mvs(const config &cfg)
{
	return new mvs(cfg);
}

}
}

