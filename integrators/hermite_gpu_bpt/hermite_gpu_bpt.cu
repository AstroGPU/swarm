#include "hip/hip_runtime.h"
#include "hermite_gpu_bpt.h"
#include "meta.hpp"

namespace swarm {
namespace hermite_gpu_bpt {

		
	__constant__ ensemble gpu_hermite_ens;

	inline __device__ static double inner_product(const double a[3],const double b[3]){
		return a[0]*b[0]+a[1]*b[1]+a[2]*b[2];
	}


	/**!
	 * helper function for accjerk_updater that operates on each component
	 * it gets scalar part of acceleration as input and calculates one component of
	 * acceleration and jerk at a time
	 *
	 */
	__device__ static void accjerk_updater_component(int c
			,double dx[3],double dv[3],double scalar,double rv
			,double (&acc)[3],double (&jerk)[3]){
		acc[c] += dx[c]* scalar;
		jerk[c] += (dv[c] - dx[c] * rv) * scalar;

	}

	/*! 
	 * templatized function object to calculate acceleration and jerk
	 * It updates accleration and jerk for one body: bodid. this function
	 * object is body of a n*n loop. so it should get called for every pair
	 *
	 */
	template<int nbod>
	struct accjerk_updater {
		ensemble::systemref& sysref;
		const double (&pos)[3][nbod],(&vel)[3][nbod];
		double (&acc)[3], (&jerk)[3];
		const int i;
		__device__ accjerk_updater(const int bodid,ensemble::systemref& sysref,const double (&pos)[3][nbod],const double (&vel)[3][nbod], double (&acc)[3], double (&jerk)[3])
			:sysref(sysref),pos(pos),vel(vel),acc(acc),jerk(jerk),i(bodid){
				acc[0] = acc[1] = acc[2] = 0.0;
				jerk[0] = jerk[1] = jerk[2] = 0.0;
			}
		__device__ void operator()(int j)const{
			if(i != j){

				double dx[3] =  { pos[0][j]-pos[0][i],pos[1][j]-pos[1][i],pos[2][j]-pos[2][i]};
				double dv[3] =  { vel[0][j]-vel[0][i],vel[1][j]-vel[1][i],vel[2][j]-vel[2][i]};

				// computing scalar part of the acceleration
				double r2 =  dx[0]*dx[0] + dx[1]*dx[1] + dx[2]*dx[2] ;
				double rv =  inner_product(dx,dv) * 3 / r2;
				double rinv = rsqrt(r2)  / r2;

				// vectorized part
				const double scalar_i = +rinv*sysref[j].mass();
				accjerk_updater_component(0,dx,dv,scalar_i,rv,acc,jerk);
				accjerk_updater_component(1,dx,dv,scalar_i,rv,acc,jerk);
				accjerk_updater_component(2,dx,dv,scalar_i,rv,acc,jerk);

			}


		}
	};


	template<int nbod>
		__device__ static void predictor(int i,int c,const int& s, const int& d
				,double (&pos)[2][3][nbod],double (&vel)[2][3][nbod],double (&acc)[2][3],double (&jerk)[2][3]
				,double h){
			pos[d][c][i] = pos[s][c][i] +  h*(vel[s][c][i]+(h/2)*(acc[s][c]+(h/3)*jerk[s][c]));
			vel[d][c][i] = vel[s][c][i] +  h*(acc[s][c]+(h/2)*jerk[s][c]);
		}

	template<int nbod>
	__device__ static void corrector(int i,const int& c,const int& s, const int& d
			,double (&pos)[2][3][nbod],double (&vel)[2][3][nbod],double (&acc)[2][3],double (&jerk)[2][3]
			,const double& h){
		pos[d][c][i] = pos[s][c][i] + (h/2) * ( (vel[s][c][i]+vel[d][c][i]) 
				+ (h*7.0/30)*( (acc[s][c]-acc[d][c]) + (h/7) * (jerk[s][c]+jerk[d][c])));
		vel[d][c][i] = vel[s][c][i] + (h/2) * ( (acc[s][c]+acc[d][c]) + (h/6) * (jerk[s][c]-jerk[d][c]));
	}

	__device__ void copy(double src[3],double des[3]){
		des[0] = src[0], des[1] = src[1] , des[2] = src[2];
	}

	template<int nbod>
	__device__ void load_to_shared(double pos[2][3][nbod],double vel[2][3][nbod],ensemble::systemref& sysref,int k,int c,int i){
		pos[k][c][i] = sysref[i].p(c), vel[k][c][i] = sysref[i].v(c);
	}

	template<int nbod>
	__device__ void store_from_shared(double pos[2][3][nbod],double vel[2][3][nbod],ensemble::systemref& sysref,int k,int c,int i){
		sysref[i].p(c) = pos[k][c][i] ,  sysref[i].v(c) = vel[k][c][i];
	}


	template<int nbod>
	__global__ void gpu_hermite_bpt_integrator_kernel(double destination_time, double time_step){
		
		// this kernel will process specific body of specific system
		// getting essential pointers
		int sysid = ((blockIdx.z * gridDim.y + blockIdx.y) * gridDim.x + blockIdx.x) * blockDim.y + threadIdx.y;
		int sysid_in_block = threadIdx.y;
		int bodid = threadIdx.x;
		ensemble &ens = gpu_hermite_ens;
		if(sysid >= ens.nsys() || bodid >= nbod) { return; }
		ensemble::systemref gsys ( ens[sysid] );


		// shared memory allocation
		extern __shared__ char shared_mem[];
		double (*shared_array)[2][3][nbod] = (double (*)[2][3][nbod]) shared_mem;
			
		// pointers to shared_memory
		double (&pos)[2][3][nbod] = shared_array[sysid_in_block*2], (&vel)[2][3][nbod] = shared_array[sysid_in_block*2+1];
		// local memory allocation
		double acc[2][3], jerk[2][3] ;

		double t_start = gsys.time(), t = t_start;
		double t_end = min(t_start + destination_time,gsys.time_end());

		// Load data into shared memory (cooperative load)
		load_to_shared<nbod>(pos,vel,gsys,0,0,bodid);
		load_to_shared<nbod>(pos,vel,gsys,0,1,bodid);
		load_to_shared<nbod>(pos,vel,gsys,0,2,bodid);
		
		__syncthreads(); // load should complete before calculating acceleration and jerk

		// Calculate acceleration and jerk
		Unroller<0,nbod>::step(accjerk_updater<nbod>(bodid,gsys,pos[0],vel[0],acc[0],jerk[0]));

		while(t < t_end){

			{
				double h = min(time_step, t_end - t);
				// these two variable determine how each half of pos/vel/acc/jerk arrays
				// are going to be used to avoid unnecessary copying.
				const int s = 0, d = 1; 
				// Predict 
				predictor<nbod>(bodid,0,s,d,pos,vel,acc,jerk,h);
				predictor<nbod>(bodid,1,s,d,pos,vel,acc,jerk,h);
				predictor<nbod>(bodid,2,s,d,pos,vel,acc,jerk,h);


				// Do evaluation and correction two times (PEC2)
				{
					__syncthreads();
					// Calculate acceleration and jerk
					accjerk_updater<nbod> accjerk_updater_instance(bodid,gsys,pos[d],vel[d],acc[d],jerk[d]);
					Unroller<0,nbod>::step(accjerk_updater_instance);

					//__syncthreads(); // to prevent WAR. corrector updates pos/vel that accjerk_updater would read

					// Correct
					corrector<nbod>(bodid,0,s,d,pos,vel,acc,jerk,h);
					corrector<nbod>(bodid,1,s,d,pos,vel,acc,jerk,h);
					corrector<nbod>(bodid,2,s,d,pos,vel,acc,jerk,h);

				}
				{
					__syncthreads();
					// Calculate acceleration and jerk
					accjerk_updater<nbod> accjerk_updater_instance(bodid,gsys,pos[d],vel[d],acc[d],jerk[d]);
					Unroller<0,nbod>::step(accjerk_updater_instance);

					//__syncthreads(); // to prevent WAR. corrector updates pos/vel that accjerk_updater would read

					// Correct
					corrector<nbod>(bodid,0,s,d,pos,vel,acc,jerk,h);
					corrector<nbod>(bodid,1,s,d,pos,vel,acc,jerk,h);
					corrector<nbod>(bodid,2,s,d,pos,vel,acc,jerk,h);

				}

				t += h;

			}
			if(bodid == 0) 
				gsys.increase_stepcount();
			// the following block is exact copy of block above with only change in s,d
			// please don't edit and always copy from block above
			{
				double h = min(time_step, t_end - t);
				// these two variable determine how each half of pos/vel/acc/jerk arrays
				// are going to be used to avoid unnecessary copying.
				const int s = 1, d = 0; 
				// Predict 
				predictor<nbod>(bodid,0,s,d,pos,vel,acc,jerk,h);
				predictor<nbod>(bodid,1,s,d,pos,vel,acc,jerk,h);
				predictor<nbod>(bodid,2,s,d,pos,vel,acc,jerk,h);


				// Do evaluation and correction two times (PEC2)
				{
					__syncthreads();
					// Calculate acceleration and jerk
					accjerk_updater<nbod> accjerk_updater_instance(bodid,gsys,pos[d],vel[d],acc[d],jerk[d]);
					Unroller<0,nbod>::step(accjerk_updater_instance);

					//__syncthreads(); // to prevent WAR. corrector updates pos/vel that accjerk_updater would read

					// Correct
					corrector<nbod>(bodid,0,s,d,pos,vel,acc,jerk,h);
					corrector<nbod>(bodid,1,s,d,pos,vel,acc,jerk,h);
					corrector<nbod>(bodid,2,s,d,pos,vel,acc,jerk,h);

				}
				{
					__syncthreads();
					// Calculate acceleration and jerk
					accjerk_updater<nbod> accjerk_updater_instance(bodid,gsys,pos[d],vel[d],acc[d],jerk[d]);
					Unroller<0,nbod>::step(accjerk_updater_instance);

					//__syncthreads(); // to prevent WAR. corrector updates pos/vel that accjerk_updater would read

					// Correct
					corrector<nbod>(bodid,0,s,d,pos,vel,acc,jerk,h);
					corrector<nbod>(bodid,1,s,d,pos,vel,acc,jerk,h);
					corrector<nbod>(bodid,2,s,d,pos,vel,acc,jerk,h);

				}

				t += h;

			}
			debug_hook();

			if(bodid == 0) 
				gsys.increase_stepcount();

			if(log::needs_output(ens, t, sysid))
			{
				// Save pos/vel to global memory
				store_from_shared<nbod>(pos,vel,gsys,0,0,bodid);
				store_from_shared<nbod>(pos,vel,gsys,0,1,bodid);
				store_from_shared<nbod>(pos,vel,gsys,0,2,bodid);
				if(bodid == 0) {
					gsys.set_time(t);
					log::output_system(dlog, ens, t, sysid);
				}
			}

		}

		if(bodid == 0) 
			gsys.set_time(t);
		// Save pos/vel to global memory
		store_from_shared<nbod>(pos,vel,gsys,0,0,bodid);
		store_from_shared<nbod>(pos,vel,gsys,0,1,bodid);
		store_from_shared<nbod>(pos,vel,gsys,0,2,bodid);

	}

	//! Simple template Function Object to execute appropriate kernel at runtime
	template<int nbod>
		struct kernel_launcher {
			template<class P>
				static void choose(P p){
					double dT = p.dT;
					double h = p.h;
					gpu_hermite_bpt_integrator_kernel<nbod><<<p.gridDim, p.threadDim, p.shared_memory_size>>>(dT, h);
				}
		};



	inline int min_power_2(const int& x){
		int y;
		for(y = 1; y < x; y*=2);
		return y;
	}
	/*!
	 * \brief host function to invoke a kernel (double precision) 
	 *
	 * Currently maximum number of bodies is set to 10.
	 * In order to change, add if statement. 
	 * @param[in,out] ens gpu_ensemble for data communication
	 * @param[in] dT destination time 
	 */
		void gpu_hermite_bpt_integrator::integrate(gpu_ensemble &ens, double dT)
		{
			/* Upload the kernel parameters */ 
			if(ens.last_integrator() != this) 
			{ 
				int system_per_block = threadsPerBlock / ens.nbod();
				threadDim.x = ens.nbod();
				threadDim.y = system_per_block;

				this->shared_memory_size = system_per_block * ens.nbod() * 2 * 2 * 3 * sizeof(double);

				ens.set_last_integrator(this); 
				configure_grid(gridDim,  system_per_block , ens.nsys()); 
				hipMemcpyToSymbol(HIP_SYMBOL(gpu_hermite_ens),	&ens, sizeof(gpu_hermite_ens) ); 
				if(dT == 0.) { return; } 
			} 
			// flush CPU/GPU output logs
			log::flush(log::memory | log::if_full);

			this->dT = dT;
			const int MAX_NBODIES = 10;
			if(ens.nbod() <= MAX_NBODIES){
				choose<kernel_launcher,3,MAX_NBODIES,void>(ens.nbod(),*this);
			} else {
				// How do we get an error message out of here?
				ERROR("Invalid number of bodies. (only up to 10 bodies per system)");
				return;
			}
			// flush CPU/GPU output logs
			log::flush(log::memory);

		}

}
}
