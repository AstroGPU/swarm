#include "hip/hip_runtime.h"
#include "swarm.h"
#include "hermite_adap_gpu.h"

//#if PRECISION == 3
// Double precision
#define RSQRT(x) rsqrtf(x)
#define SQRT(x)   sqrtf(x)
//typedef double real;
//#else


namespace gpu_hermite_adap_aux
{
	//
	// Wrap all aux. functions in a separate namespace, to avoid
	// collisions with equally named functions from other integrators.
	//

	__device__ float3 operator*(const float3 &a, const float &b)
	{
		return make_float3(a.x*b, a.y*b, a.z*b);
	}

	__device__ float3 operator+(const float3 &a, const float3 &b)
	{
		return make_float3(a.x+b.x, a.y+b.y, a.z+b.z);
	}


inline __device__ real getAdaptiveTimeStep(real *mAcc, real *mJerk, unsigned int nBodies, real h) 
 {
         real dt=LARGE_NUMBER;
         real magAcc, magJerk;
         for(unsigned int i=0;i<nBodies;++i) 
          {
                   magAcc =sqrt(pow(mAcc[i*3],2) +pow(mAcc[i*3+1],2) +pow(mAcc[i*3+2],2) ) ;
                   magJerk=sqrt(pow(mJerk[i*3],2)+pow(mJerk[i*3+1],2)+pow(mJerk[i*3+2],2) ) ;
                   dt=min(dt,magAcc/(magJerk+magAcc*SMALL_NUMBER)*STEP_FACTOR+h);
          }
         return dt;
 }


template<unsigned int N, typename destT, typename srcT>
inline __device__ void copyArray(destT *target, srcT *source)
{
	if(N==9){
		target[0]=source[0];
		target[1]=source[1];
		target[2]=source[2];
		target[3]=source[3];
		target[4]=source[4];
		target[5]=source[5];
		target[6]=source[6];
		target[7]=source[7];
		target[8]=source[8];
	}
	else {
		for(int i=0; i<N; i++)
			target[i]=source[i];
	}
}

template<unsigned int N> 
inline __device__ void doubleTofloat(float *floatA, real *doubleA)
{
	copyArray<N,float,real>(floatA,doubleA);
}

template<unsigned int N>
inline __device__ void floatTodouble(double *doubleA, float *floatA)
{
	copyArray<N,real,float>(doubleA,floatA);
}

inline __device__ void predict(real *mPos, real *mVel, real *mAcc, real *mJerk, const real dtby2, const real dtby3, double h, int numArray)
{
	if(numArray==9){
		mPos[0] += h* (mVel[0]+ dtby2*(mAcc[0]+dtby3*mJerk[0]));
		mPos[1] += h* (mVel[1]+ dtby2*(mAcc[1]+dtby3*mJerk[1]));
		mPos[2] += h* (mVel[2]+ dtby2*(mAcc[2]+dtby3*mJerk[2]));
		mPos[3] += h* (mVel[3]+ dtby2*(mAcc[3]+dtby3*mJerk[3]));
		mPos[4] += h* (mVel[4]+ dtby2*(mAcc[4]+dtby3*mJerk[4]));
		mPos[5] += h* (mVel[5]+ dtby2*(mAcc[5]+dtby3*mJerk[5]));
		mPos[6] += h* (mVel[6]+ dtby2*(mAcc[6]+dtby3*mJerk[6]));
		mPos[7] += h* (mVel[7]+ dtby2*(mAcc[7]+dtby3*mJerk[7]));
		mPos[8] += h* (mVel[8]+ dtby2*(mAcc[8]+dtby3*mJerk[8]));
		mVel[0] += h* (mAcc[0]+ dtby2*mJerk[0]);
		mVel[1] += h* (mAcc[1]+ dtby2*mJerk[1]);
		mVel[2] += h* (mAcc[2]+ dtby2*mJerk[2]);
		mVel[3] += h* (mAcc[3]+ dtby2*mJerk[3]);
		mVel[4] += h* (mAcc[4]+ dtby2*mJerk[4]);
		mVel[5] += h* (mAcc[5]+ dtby2*mJerk[5]);
		mVel[6] += h* (mAcc[6]+ dtby2*mJerk[6]);
		mVel[7] += h* (mAcc[7]+ dtby2*mJerk[7]);
		mVel[8] += h* (mAcc[8]+ dtby2*mJerk[8]);
	}
	else {
		for(int i=0; i<numArray; i++) {
			mPos[i] += h* (mVel[i]+ dtby2*(mAcc[i]+dtby3*mJerk[i]));
			mVel[i] += h* (mAcc[i]+ dtby2*mJerk[i]);
		}
	}
}

inline __device__ void correct(real *mPos, real *mVel, real *mAcc, real *mJerk, 
		real *mPosOld, real *mVelOld, real *mAccOld, real *mJerkOld, 
		const real dtby2, const real dtby6, const real dtby7, const real dt7by30, int numArray)
{
	if(numArray==9){
		mVel[0] = mVelOld[0] + dtby2*((mAccOld[0]+mAcc[0]) + dtby6*  (mJerkOld[0]-mJerk[0]));
		mVel[1] = mVelOld[1] + dtby2*((mAccOld[1]+mAcc[1]) + dtby6*  (mJerkOld[1]-mJerk[1]));
		mVel[2] = mVelOld[2] + dtby2*((mAccOld[2]+mAcc[2]) + dtby6*  (mJerkOld[2]-mJerk[2]));
		mVel[3] = mVelOld[3] + dtby2*((mAccOld[3]+mAcc[3]) + dtby6*  (mJerkOld[3]-mJerk[3]));
		mVel[4] = mVelOld[4] + dtby2*((mAccOld[4]+mAcc[4]) + dtby6*  (mJerkOld[4]-mJerk[4]));
		mVel[5] = mVelOld[5] + dtby2*((mAccOld[5]+mAcc[5]) + dtby6*  (mJerkOld[5]-mJerk[5]));
		mVel[6] = mVelOld[6] + dtby2*((mAccOld[6]+mAcc[6]) + dtby6*  (mJerkOld[6]-mJerk[6]));
		mVel[7] = mVelOld[7] + dtby2*((mAccOld[7]+mAcc[7]) + dtby6*  (mJerkOld[7]-mJerk[7]));
		mVel[8] = mVelOld[8] + dtby2*((mAccOld[8]+mAcc[8]) + dtby6*  (mJerkOld[8]-mJerk[8]));
		mPos[0] = mPosOld[0] + dtby2*((mVelOld[0]+mVel[0]) + dt7by30*((mAccOld[0]- mAcc[0]) + dtby7*(mJerkOld[0]+mJerk[0])));
		mPos[1] = mPosOld[1] + dtby2*((mVelOld[1]+mVel[1]) + dt7by30*((mAccOld[1]- mAcc[1]) + dtby7*(mJerkOld[1]+mJerk[1])));
		mPos[2] = mPosOld[2] + dtby2*((mVelOld[2]+mVel[2]) + dt7by30*((mAccOld[2]- mAcc[2]) + dtby7*(mJerkOld[2]+mJerk[2])));
		mPos[3] = mPosOld[3] + dtby2*((mVelOld[3]+mVel[3]) + dt7by30*((mAccOld[3]- mAcc[3]) + dtby7*(mJerkOld[3]+mJerk[3])));
		mPos[4] = mPosOld[4] + dtby2*((mVelOld[4]+mVel[4]) + dt7by30*((mAccOld[4]- mAcc[4]) + dtby7*(mJerkOld[4]+mJerk[4])));
		mPos[5] = mPosOld[5] + dtby2*((mVelOld[5]+mVel[5]) + dt7by30*((mAccOld[5]- mAcc[5]) + dtby7*(mJerkOld[5]+mJerk[5])));
		mPos[6] = mPosOld[6] + dtby2*((mVelOld[6]+mVel[6]) + dt7by30*((mAccOld[6]- mAcc[6]) + dtby7*(mJerkOld[6]+mJerk[6])));
		mPos[7] = mPosOld[7] + dtby2*((mVelOld[7]+mVel[7]) + dt7by30*((mAccOld[7]- mAcc[7]) + dtby7*(mJerkOld[7]+mJerk[7])));
		mPos[8] = mPosOld[8] + dtby2*((mVelOld[8]+mVel[8]) + dt7by30*((mAccOld[8]- mAcc[8]) + dtby7*(mJerkOld[8]+mJerk[8])));
	}
	else {
		for(int i=0; i<numArray; i++) {
			mVel[i] = mVelOld[i] + dtby2*((mAccOld[i]+mAcc[i]) + dtby6*  (mJerkOld[i]-mJerk[i]));
			mPos[i] = mPosOld[i] + dtby2*((mVelOld[i]+mVel[i]) + dt7by30*((mAccOld[i]- mAcc[i]) + dtby7*(mJerkOld[i]+mJerk[i])));
		}
	}
}
//******************************************************************
// * UpdateAccJerk function for 2 or 3 Planets 
// *(real = float for single and mixed)
// *(real = double for double)
//******************************************************************
template<class acc_real>
__device__  void UpdateAccJerk(acc_real * mPos, acc_real * mVel, acc_real* mAcc, acc_real* mJerk, int nBodies,const float * d_mass) 
{
	acc_real dx[]={0,0,0}; 
	acc_real dv[]={0,0,0}; 
	acc_real dx_back[]={0,0,0}; 
	acc_real dv_back[]={0,0,0}; 

	acc_real r2=0;
	acc_real rv=0;
	acc_real rinv=0;
	acc_real rinv3=0;

	acc_real ai0[]={0,0,0};
	acc_real ai1[]={0,0,0};
	acc_real ai2[]={0,0,0};
	acc_real ji0[]={0,0,0};
	acc_real ji1[]={0,0,0};
	acc_real ji2[]={0,0,0};

	//if(nBodies ==4) {
#if NBODIES>3
	acc_real ai3[]={0,0,0};
	acc_real ji3[]={0,0,0};
#endif
	//}

	//! planet1 and planet2
	dx[0] = mPos[6] - mPos[3]; dx_back[0] = -dx[0];
	dx[1] = mPos[7] - mPos[4]; dx_back[1] = -dx[1];
	dx[2] = mPos[8] - mPos[5]; dx_back[2] = -dx[2];
	dv[0] = mVel[6] - mVel[3]; dv_back[0] = -dv[0];
	dv[1] = mVel[7] - mVel[4]; dv_back[1] = -dv[1];
	dv[2] = mVel[8] - mVel[5]; dv_back[2] = -dv[2];

	r2 = dx[0]*dx[0] + dx[1]*dx[1] + dx[2]*dx[2];
	rv = dx[0]*dv[0] + dx[1]*dv[1] + dx[2]*dv[2];
	rinv = RSQRT(r2);
	rv *= 3.0f/r2;
	rinv *= d_mass[2];
	rinv3 =  rinv/r2;

	dx[0] *= rinv3; dx[1] *= rinv3; dx[2] *= rinv3;
	ai1[0] += dx[0]; ai1[1] += dx[1]; ai1[2] += dx[2];
	dv[0] *= rinv3; dv[1] *= rinv3; dv[2] *= rinv3;
	ji1[0] += dv[0]; ji1[1] += dv[1]; ji1[2] += dv[2];
	dx[0] *= rv; dx[1] *= rv; dx[2] *= rv;
	ji1[0] -= dx[0]; ji1[1] -= dx[1]; ji1[2] -= dx[2];


	rinv3 = rinv3/d_mass[2] * d_mass[1];

	dx_back[0] *= rinv3; dx_back[1] *= rinv3; dx_back[2] *= rinv3;
	ai2[0] += dx_back[0]; ai2[1] += dx_back[1]; ai2[2] += dx_back[2];
	dv_back[0] *= rinv3; dv_back[1] *= rinv3; dv_back[2] *= rinv3;
	ji2[0] += dv_back[0]; ji2[1] += dv_back[1]; ji2[2] += dv_back[2];
	ji2[0] -= dx_back[0]*rv; ji2[1] -= dx_back[1]*rv; ji2[2] -= dx_back[2]*rv;

#if NBODIES>3
		//! planet1 and planet 3
		dx[0] = mPos[9] - mPos[3]; dx_back[0] = -dx[0];
		dx[1] = mPos[10] - mPos[4]; dx_back[1] = -dx[1];
		dx[2] = mPos[11] - mPos[5]; dx_back[2] = -dx[2];
		dv[0] = mVel[9] - mVel[3]; dv_back[0] = -dv[0];
		dv[1] = mVel[10] - mVel[4]; dv_back[1] = -dv[1];
		dv[2] = mVel[11] - mVel[5]; dv_back[2] = -dv[2];

		r2 = dx[0]*dx[0] + dx[1]*dx[1] + dx[2]*dx[2];
		rv = dx[0]*dv[0] + dx[1]*dv[1] + dx[2]*dv[2];
		rinv = RSQRT(r2);
		rv *= 3.0f/r2;
		rinv *= d_mass[3];
		rinv3 = rinv/r2;

		dx[0] *= rinv3; dx[1] *= rinv3; dx[2] *= rinv3;
		ai1[0] += dx[0]; ai1[1] += dx[1]; ai1[2] += dx[2];
		dv[0] *= rinv3; dv[1] *= rinv3; dv[2] *= rinv3;
		ji1[0] += dv[0]; ji1[1] += dv[1]; ji1[2] += dv[2];
		dx[0] *= rv; dx[1] *= rv; dx[2] *= rv;
		ji1[0] -= dx[0]; ji1[1] -= dx[1]; ji1[2] -= dx[2];


		rinv3 = rinv3/d_mass[3] * d_mass[1];

		dx_back[0] *= rinv3; dx_back[1] *= rinv3; dx_back[2] *= rinv3;
		ai3[0] += dx_back[0]; ai3[1] += dx_back[1]; ai3[2] += dx_back[2];
		dv_back[0] *= rinv3; dv_back[1] *= rinv3; dv_back[2] *= rinv3;
		ji3[0] += dv_back[0]; ji3[1] += dv_back[1]; ji3[2] += dv_back[2];
		dx_back[0] *= rv; dx_back[1] *= rv; dx_back[2] *= rv;
		ji3[0] -= dx_back[0]; ji3[1] -= dx_back[1]; ji3[2] -= dx_back[2];

		//! planet2 and planet 3
		dx[0] = mPos[9] - mPos[6]; dx_back[0] = -dx[0];
		dx[1] = mPos[10] - mPos[7]; dx_back[1] = -dx[1];
		dx[2] = mPos[11] - mPos[8]; dx_back[2] = -dx[2];
		dv[0] = mVel[9] - mVel[6]; dv_back[0] = -dv[0];
		dv[1] = mVel[10] - mVel[7]; dv_back[1] = -dv[1];
		dv[2] = mVel[11] - mVel[8]; dv_back[2] = -dv[2];


		r2 = dx[0]*dx[0] + dx[1]*dx[1] + dx[2]*dx[2];
		rv = dx[0]*dv[0] + dx[1]*dv[1] + dx[2]*dv[2];
		rinv = RSQRT(r2);
		rv *= 3.0f/r2;
		rinv *= d_mass[3];
		rinv3 = rinv/r2;

		dx[0] *= rinv3; dx[1] *= rinv3; dx[2] *= rinv3;
		ai2[0] += dx[0]; ai2[1] += dx[1]; ai2[2] += dx[2];
		dv[0] *= rinv3; dv[1] *= rinv3; dv[2] *= rinv3;
		ji2[0] += dv[0]; ji2[1] += dv[1]; ji2[2] += dv[2];
		dx[0] *= rv; dx[1] *= rv; dx[2] *= rv;
		ji2[0] -= dx[0]; ji2[1] -= dx[1]; ji2[2] -= dx[2];

		rinv3 = rinv3/d_mass[3] * d_mass[2];

		dx_back[0] *= rinv3; dx_back[1] *= rinv3; dx_back[2] *= rinv3;
		ai3[0] += dx_back[0]; ai3[1] += dx_back[1]; ai3[2] += dx_back[2];
		dv_back[0] *= rinv3; dv_back[1] *= rinv3; dv_back[2] *= rinv3;
		ji3[0] += dv_back[0]; ji3[1] += dv_back[1]; ji3[2] += dv_back[2];
		dx_back[0] *= rv; dx_back[1] *= rv; dx_back[2] *= rv;
		ji3[0] -= dx_back[0]; ji3[1] -= dx_back[1]; ji3[2] -= dx_back[2];
#endif
	
	//! Star and planet 1
	dx[0] = mPos[0] - mPos[3]; dx_back[0] = -dx[0];
	dx[1] = mPos[1] - mPos[4]; dx_back[1] = -dx[1];
	dx[2] = mPos[2] - mPos[5]; dx_back[2] = -dx[2];
	dv[0] = mVel[0] - mVel[3]; dv_back[0] = -dv[0];
	dv[1] = mVel[1] - mVel[4]; dv_back[1] = -dv[1];
	dv[2] = mVel[2] - mVel[5]; dv_back[2] = -dv[2];


	r2 = dx[0]*dx[0] + dx[1]*dx[1] + dx[2]*dx[2];
	rv = dx[0]*dv[0] + dx[1]*dv[1] + dx[2]*dv[2];
	rinv = RSQRT(r2);
	rv *= 3.0f/r2;
	rinv *= d_mass[0];
	rinv3 = rinv/r2;

	dx[0] *= rinv3; dx[1] *= rinv3; dx[2] *= rinv3;
	ai1[0] += dx[0]; ai1[1] += dx[1]; ai1[2] += dx[2];
	dv[0] *= rinv3; dv[1] *= rinv3; dv[2] *= rinv3;
	ji1[0] += dv[0]; ji1[1] += dv[1]; ji1[2] += dv[2];
	dx[0] *= rv; dx[1] *= rv; dx[2] *= rv;
	ji1[0] -= dx[0]; ji1[1] -= dx[1]; ji1[2] -= dx[2];


	rinv3=rinv3/d_mass[0]*d_mass[1];

	dx_back[0] *= rinv3; dx_back[1] *= rinv3; dx_back[2] *= rinv3;
	ai0[0] += dx_back[0]; ai0[1] += dx_back[1]; ai0[2] += dx_back[2];
	dv_back[0] *= rinv3; dv_back[1] *= rinv3; dv_back[2] *= rinv3;
	ji0[0] += dv_back[0]; ji0[1] += dv_back[1]; ji0[2] += dv_back[2];
	dx_back[0] *= rv; dx_back[1] *= rv; dx_back[2] *= rv;
	ji0[0] -= dx_back[0]; ji0[1] -= dx_back[1]; ji0[2] -= dx_back[2];


	//! Star and planet 2
	dx[0] = mPos[6] - mPos[0]; dx_back[0] = -dx[0];
	dx[1] = mPos[7] - mPos[1]; dx_back[1] = -dx[1];
	dx[2] = mPos[8] - mPos[2]; dx_back[2] = -dx[2];
	dv[0] = mVel[6] - mVel[0]; dv_back[0] = -dv[0];
	dv[1] = mVel[7] - mVel[1]; dv_back[1] = -dv[1];
	dv[2] = mVel[8] - mVel[2]; dv_back[2] = -dv[2];

	r2 = dx[0]*dx[0] + dx[1]*dx[1] + dx[2]*dx[2];
	rv = dx[0]*dv[0] + dx[1]*dv[1] + dx[2]*dv[2];
	rinv = RSQRT(r2);
	rv *= 3.0f/r2;
	rinv *= d_mass[2];
	rinv3 = rinv/r2;

	dx[0] *= rinv3; dx[1] *= rinv3; dx[2] *= rinv3;
	ai0[0] += dx[0]; ai0[1] += dx[1]; ai0[2] += dx[2];
	dv[0] *= rinv3; dv[1] *= rinv3; dv[2] *= rinv3;
	ji0[0] += dv[0]; ji0[1] += dv[1]; ji0[2] += dv[2];
	dx[0] *= rv; dx[1] *= rv; dx[2] *= rv;
	ji0[0] -= dx[0]; ji0[1] -= dx[1]; ji0[2] -= dx[2];

	rinv3 = rinv3/d_mass[2] * d_mass[0];

	dx_back[0] *= rinv3; dx_back[1] *= rinv3; dx_back[2] *= rinv3;
	ai2[0] += dx_back[0]; ai2[1] += dx_back[1]; ai2[2] += dx_back[2];
	dv_back[0] *= rinv3; dv_back[1] *= rinv3; dv_back[2] *= rinv3;
	ji2[0] += dv_back[0]; ji2[1] += dv_back[1]; ji2[2] += dv_back[2];
	dx_back[0] *= rv; dx_back[1] *= rv; dx_back[2] *= rv;
	ji2[0] -= dx_back[0]; ji2[1] -= dx_back[1]; ji2[2] -= dx_back[2];
#if NBODIES>3
		//! Star and planet 3
		dx[0] = mPos[9] - mPos[0]; dx_back[0] = -dx[0];
		dx[1] = mPos[10] - mPos[1]; dx_back[1] = -dx[1];
		dx[2] = mPos[11] - mPos[2]; dx_back[2] = -dx[2];
		dv[0] = mVel[9] - mVel[0]; dv_back[0] = -dv[0];
		dv[1] = mVel[10] - mVel[1]; dv_back[1] = -dv[1];
		dv[2] = mVel[11] - mVel[2]; dv_back[2] = -dv[2];

		r2 = dx[0]*dx[0] + dx[1]*dx[1] + dx[2]*dx[2];
		rv = dx[0]*dv[0] + dx[1]*dv[1] + dx[2]*dv[2];
		rinv = RSQRT(r2);
		rv *= 3.0f/r2;
		rinv *= d_mass[3];
		rinv3 = rinv/r2;

		dx[0] *= rinv3; dx[1] *= rinv3; dx[2] *= rinv3;
		ai0[0] += dx[0]; ai0[1] += dx[1]; ai0[2] += dx[2];
		dv[0] *= rinv3; dv[1] *= rinv3; dv[2] *= rinv3;
		ji0[0] += dv[0]; ji0[1] += dv[1]; ji0[2] += dv[2];
		dx[0] *= rv; dx[1] *= rv; dx[2] *= rv;
		ji0[0] -= dx[0]; ji0[1] -= dx[1]; ji0[2] -= dx[2];

		rinv3 = rinv3/d_mass[3] * d_mass[0];

		dx_back[0] *= rinv3; dx_back[1] *= rinv3; dx_back[2] *= rinv3;
		ai3[0] += dx_back[0]; ai3[1] += dx_back[1]; ai3[2] += dx_back[2];
		dv_back[0] *= rinv3; dv_back[1] *= rinv3; dv_back[2] *= rinv3;
		ji3[0] += dv_back[0]; ji3[1] += dv_back[1]; ji3[2] += dv_back[2];
		dx_back[0] *= rv; dx_back[1] *= rv; dx_back[2] *= rv;
		ji3[0] -= dx_back[0]; ji3[1] -= dx_back[1]; ji3[2] -= dx_back[2];

		mAcc[9] = ai3[0]; mAcc[10] = ai3[1]; mAcc[11] = ai3[2]; 
		mJerk[9] = ji3[0]; mJerk[10] = ji3[1]; mJerk[11] = ji3[2];
#endif
	
	mAcc[0] = ai0[0]; mAcc[1] = ai0[1]; mAcc[2] = ai0[2]; 
	mJerk[0] = ji0[0]; mJerk[1] = ji0[1]; mJerk[2] = ji0[2];
	mAcc[3] = ai1[0]; mAcc[4] = ai1[1]; mAcc[5] = ai1[2]; 
	mJerk[3] = ji1[0]; mJerk[4] = ji1[1]; mJerk[5] = ji1[2];
	mAcc[6] = ai2[0]; mAcc[7] = ai2[1]; mAcc[8] = ai2[2]; 
	mJerk[6] = ji2[0]; mJerk[7] = ji2[1]; mJerk[8] = ji2[2];
}

//******************************************************************
// * UpdateAccJerk function for more than 3 Planets 
// *(real = float for single and mixed)
// *(real = double for double)
// ******************************************************************/
template<class acc_real>
__device__  void UpdateAccJerk_General(acc_real * mPos, acc_real * mVel, acc_real* mAcc, acc_real* mJerk, int nBodies,const float * d_mass) 
{

	real dx[]={0,0,0}; 
	real dv[]={0,0,0}; 

	{ // First calculate acceleration and jerk for the Sun
		unsigned int i = 0;
		real xi[]={mPos[i*3], mPos[i*3+1], mPos[i*3+2]};
		real vi[]={mVel[i*3], mVel[i*3+1], mVel[i*3+2]};
		real ai[]={0,0,0};
		real ji[]={0,0,0};

#pragma unroll
		for(unsigned int j=1;j<nBodies;++j)
		{
			unsigned int jj= 3*j;
			dx[0] = mPos[jj] - xi[0];
			dv[0] = mVel[jj] - vi[0];
			++jj;
			dx[1] = mPos[jj] - xi[1];
			dv[1] = mVel[jj] - vi[1];
			++jj;
			dx[2] = mPos[jj] - xi[2];
			dv[2] = mVel[jj] - vi[2];
			real r2 = dx[0]*dx[0] + dx[1]*dx[1] + dx[2]*dx[2];
			real rv = dx[0]*dv[0] + dx[1]*dv[1] + dx[2]*dv[2];
			real rinv = RSQRT(r2);
			rv *= 3./r2;
			rinv *= d_mass[j];
			real rinv3 = rinv/r2;

			//dx *= rinv3;
			//dx = rinv3*dx;
			dx[0] *= rinv3;
			dx[1] *= rinv3;
			dx[2] *= rinv3;
			//ai += dx;
			ai[0] += dx[0];
			ai[1] += dx[1];
			ai[2] += dx[2];
			//dv *= rinv3;
			//dv = rinv3*dv;
			dv[0] *= rinv3;
			dv[1] *= rinv3;
			dv[2] *= rinv3;
			//ji += dv;
			ji[0] += dv[0];
			ji[1] += dv[1];
			ji[2] += dv[2];
			//dx *= rv;
			//dx = rv*dx;
			dx[0] *= rv;
			dx[1] *= rv;
			dx[2] *= rv;
			//ji -= dx;
			ji[0] -= dx[0];
			ji[1] -= dx[1];
			ji[2] -= dx[2];
		}
		//mAcc[i] = ai;
		mAcc[i*3  ] = ai[0];
		mAcc[i*3+1] = ai[1];
		mAcc[i*3+2] = ai[2];
		//mJerk[i] = ji;
		mJerk[i*3  ] = ji[0];
		mJerk[i*3+1] = ji[1];
		mJerk[i*3+2] = ji[2];
		unsigned int ii = i*3;
		mAcc[ii  ] = ai[0];
		mJerk[ii ] = ji[0];
		++ii;
		mAcc[ii ] = ai[1];
		mJerk[ii] = ji[1];
		++ii;
		mAcc[ii ] = ai[2];
		mJerk[ii] = ji[2];
	}

#pragma unroll 
	for(unsigned int i=1;i<nBodies;++i)
	{
		//float3 xi=mPos[i];
		real xi[]={mPos[i*3], mPos[i*3+1], mPos[i*3+2]};
		real vi[]={mVel[i*3], mVel[i*3+1], mVel[i*3+2]};
		real ai[]={0,0,0};
		real ji[]={0,0,0};

#pragma unroll
		for(unsigned int j=1;j<nBodies;++j)
		{
			if(j==i) continue; // Ignore body interacting with itself
			unsigned int jj= 3*j;
			dx[0] = mPos[jj] - xi[0]; dv[0] = mVel[jj] - vi[0]; ++jj;
			dx[1] = mPos[jj] - xi[1]; dv[1] = mVel[jj] - vi[1]; ++jj;
			dx[2] = mPos[jj] - xi[2]; dv[2] = mVel[jj] - vi[2];
			//	    dx = mPos[j] - mPos[i];
			//dx = mPos[j] - xi;
			//	    dv = mVel[j] - mVel[i];
			//dv = mVel[j] - vi;
			//float r2 = dx.MagnitudeSquared();
			real r2 = dx[0]*dx[0] + dx[1]*dx[1] + dx[2]*dx[2];
			//float r2 = dot(dx,dx);
			real rv = dx[0]*dv[0] + dx[1]*dv[1] + dx[2]*dv[2];
			//float rv = dot(dx,dv);
			real rinv = RSQRT(r2);
			rv *= 3./r2;
			rinv *= d_mass[j];
			real rinv3 = rinv/r2;

			//dx *= rinv3;
			//dx = rinv3*dx;
			dx[0] = rinv3*dx[0]; dx[1] = rinv3*dx[1]; dx[2] = rinv3*dx[2];
			//ai += dx;
			ai[0] = ai[0] +dx[0]; ai[1] = ai[1] +dx[1]; ai[2] = ai[2] +dx[2];
			//dv *= rinv3;
			//dv = rinv3*dv;
			dv[0] = rinv3*dv[0]; dv[1] = rinv3*dv[1]; dv[2] = rinv3*dv[2];
			//ji += dv;
			ji[0] =ji[0] + dv[0]; ji[1] =ji[1] + dv[1]; ji[2] =ji[2] + dv[2];
			//dx *= rv;
			//dx = rv*dx;
			dx[0] = rv*dx[0]; dx[1] = rv*dx[1]; dx[2] = rv*dx[2];
			//ji -= dx;
			ji[0] = ji[0] - dx[0]; ji[1] = ji[1] - dx[1]; ji[2] = ji[2] - dx[2];
		}
		{  // But add sun's contribution last to minimize round-off error
			//	    dx = mPos[j] - mPos[i];
			dx[0] = mPos[0] - xi[0]; dx[1] = mPos[1] - xi[1]; dx[2] = mPos[2] - xi[2];
			//	    dv = mVel[j] - mVel[i];
			//dv = mVel[j] - vi;
			dv[0] = mVel[0] - vi[0]; dv[1] = mVel[1] - vi[1]; dv[2] = mVel[2] - vi[2];
			//float r2 = dx.MagnitudeSquared();
			real r2 = dx[0]*dx[0] + dx[1]*dx[1] + dx[2]*dx[2];
			//float r2 = dot(dx,dx);
			real rv = dx[0]*dv[0] + dx[1]*dv[1] + dx[2]*dv[2];
			//float rv = dot(dx,dv);
			real rinv = RSQRT(r2);
			rv *= 3./r2;
			rinv *= d_mass[0];
			real rinv3 = rinv/r2;

			//dx *= rinv3;
			//dx = rinv3*dx;
			dx[0] = rinv3*dx[0]; dx[1] = rinv3*dx[1]; dx[2] = rinv3*dx[2];
			//ai += dx;
			ai[0] = ai[0] +dx[0]; ai[1] = ai[1] +dx[1]; ai[2] = ai[2] +dx[2];
			//dv *= rinv3;
			//dv = rinv3*dv;
			dv[0] = rinv3*dv[0]; dv[1] = rinv3*dv[1]; dv[2] = rinv3*dv[2];
			//ji += dv;
			ji[0] =ji[0] + dv[0]; ji[1] =ji[1] + dv[1]; ji[2] =ji[2] + dv[2];
			//dx *= rv;
			//dx = rv*dx;
			dx[0] = rv*dx[0]; dx[1] = rv*dx[1]; dx[2] = rv*dx[2];
			//ji -= dx;
			ji[0] = ji[0] - dx[0]; ji[1] = ji[1] - dx[1]; ji[2] = ji[2] - dx[2];
		}
		unsigned int ii = i*3;
		mAcc[ii] = ai[0]; mJerk[ii] = ji[0]; ++ii;
		mAcc[ii] = ai[1]; mJerk[ii] = ji[1]; ++ii;
		mAcc[ii] = ai[2]; mJerk[ii] = ji[2];
	}
}

} // end namespace
//__constant__ gpu_hermite_integrator_data pars;
__constant__ ensemble gpu_hermite_adap_ens;

template<int pre>
__global__ void gpu_hermite_adap_integrator_kernel(double dT, double h)
{
	using namespace gpu_hermite_adap_aux;

	ensemble &ens = gpu_hermite_adap_ens;
	int sys = threadId();
	if(sys >= ens.nsys()) { return; }

	double    T = ens.time(sys);
	double Tend = T + dT;
        double dt=h;


	const unsigned int nData=3*NBODIES;

	//const unsigned int current_id = sys; 

	real mPos       [nData];
	real mVel       [nData];
	real mAcc       [nData];
	real mJerk      [nData];
	real mPosOld    [nData];
	real mVelOld    [nData];
	real mAccOld    [nData];
	real mJerkOld   [nData];

	float sPos       [nData];
	float sVel       [nData];
	float sAcc       [nData];
	float sJerk      [nData];

	//const float s_mass[]={d_mass[t_start], d_mass[t_start+1],d_mass[t_start+2]};
	//const float s_mass[]={ens.mass(sys, 0), ens.mass(sys,1), ens.mass(sys,2)};

#if NBODIES > 4
	float s_mass[NBODIES];
        for(int i=0;i<NBODIES;++i)
        {
           s_mass[i]=ens.mass(sys,i); 
        }
        //ACB
        // set data from ensemble
        //ACB
        for(unsigned int i=0;i<NBODIES;++i)
         {
           mPos[i*3  ]   =ens.x(sys,i);
           mPos[i*3+1]   =ens.y(sys,i);
           mPos[i*3+2]   =ens.z(sys,i);
           mVel[i*3  ]   =ens.vx(sys,i);
           mVel[i*3+1]   =ens.vy(sys,i);
           mVel[i*3+2]   =ens.vz(sys,i);
         }
#else

#if NBODIES>3
	const float s_mass[]={ens.mass(sys, 0), ens.mass(sys,1), ens.mass(sys,2), ens.mass(sys,3)};
#else
	const float s_mass[]={ens.mass(sys, 0), ens.mass(sys,1), ens.mass(sys,2)};
#endif

	//load data from global memory
	mPos[0]=ens.x(sys,0);
	mPos[1]=ens.y(sys,0);
	mPos[2]=ens.z(sys,0);
	mPos[3]=ens.x(sys,1);
	mPos[4]=ens.y(sys,1);
	mPos[5]=ens.z(sys,1);
	mPos[6]=ens.x(sys,2);
	mPos[7]=ens.y(sys,2);
	mPos[8]=ens.z(sys,2);
#if NBODIES>3
	mPos[9]=ens.x(sys,3);
	mPos[10]=ens.y(sys,3);
	mPos[11]=ens.z(sys,3);
#endif
	mVel[0]=ens.vx(sys,0);
	mVel[1]=ens.vy(sys,0);
	mVel[2]=ens.vz(sys,0);
	mVel[3]=ens.vx(sys,1);
	mVel[4]=ens.vy(sys,1);
	mVel[5]=ens.vz(sys,1);
	mVel[6]=ens.vx(sys,2);
	mVel[7]=ens.vy(sys,2);
	mVel[8]=ens.vz(sys,2);
#if NBODIES>3
	mVel[9]=ens.vx(sys,3);
	mVel[10]=ens.vy(sys,3);
	mVel[11]=ens.vz(sys,3);
#endif
#endif

	if(pre==1)
#if NBODIES>4     
	        UpdateAccJerk_General<double>(&mPos[0], &mVel[0], &mAcc[0], &mJerk[0], NBODIES, &s_mass[0]);
#else
	        UpdateAccJerk<double>(&mPos[0], &mVel[0], &mAcc[0], &mJerk[0], NBODIES, &s_mass[0]);
#endif
	else
	{
		doubleTofloat<nData>(sPos, mPos);
		doubleTofloat<nData>(sVel, mVel);
#if NBODIES>4
		UpdateAccJerk_General<float>(&sPos[0], &sVel[0], &sAcc[0], &sJerk[0], NBODIES, &s_mass[0]);
#else
		UpdateAccJerk<float>(&sPos[0], &sVel[0], &sAcc[0], &sJerk[0], NBODIES, &s_mass[0]);
#endif
		floatTodouble<nData>(mAcc,sAcc);
		floatTodouble<nData>(mJerk,sJerk);
	}

	while(T<Tend)
	{
		////Evolve(DeltaT);
		//CopyToOld();
                dt=getAdaptiveTimeStep(&mAcc[0], &mJerk[0], NBODIES, h);
                if(dt+T>Tend)dt=Tend-T;
         
         	real dtby2=dt/2.;
	        real dtby3=dt/3.;
	        real dtby6=dt/6.;
	        real dt7by30=dt*7./30.;
	        real dtby7=dt*7.;

		copyArray<nData>(mPosOld,mPos);
		copyArray<nData>(mVelOld,mVel);
		copyArray<nData>(mAccOld,mAcc);
		copyArray<nData>(mJerkOld,mJerk);
		//for(unsigned int i=0; i<nData; ++i) {
		//	mPosOld[i]=mPos[i];
		//	mVelOld[i]=mVel[i];
		//	mAccOld[i]=mAcc[i];
		//	mJerkOld[i]=mJerk[i];
		//}

		predict(mPos,mVel,mAcc,mJerk, dtby2, dtby3, dt,nData);
		//for(unsigned int i=0; i<nData; ++i) {
		//	mPos[i] += h* (mVel[i]+ dtby2*(mAcc[i]+dtby3*mJerk[i]));
		//	mVel[i] += h* ( mAcc[i]+ dtby2*mJerk[i]);
		//}

		if(pre==1)
#if NBODIES>4
		        UpdateAccJerk_General<double>(&mPos[0], &mVel[0], &mAcc[0], &mJerk[0], NBODIES, &s_mass[0]);
#else
			UpdateAccJerk<double>(&mPos[0], &mVel[0], &mAcc[0], &mJerk[0], NBODIES, &s_mass[0]);
#endif
		else
		{
			doubleTofloat<nData>(sPos, mPos);
			doubleTofloat<nData>(sVel, mVel);
#if NBODIES>4
			UpdateAccJerk_General<float>(&sPos[0], &sVel[0], &sAcc[0], &sJerk[0], NBODIES, &s_mass[0]);
#else
			UpdateAccJerk<float>(&sPos[0], &sVel[0], &sAcc[0], &sJerk[0], NBODIES, &s_mass[0]);
#endif
			floatTodouble<nData>(mAcc,sAcc);
			floatTodouble<nData>(mJerk,sJerk);
		}

		//Correct(dt);
		correct(mPos,mVel,mAcc,mJerk, mPosOld,mVelOld,mAccOld,mJerkOld, dtby2, dtby6, dtby7, dt7by30, nData);
		
		if(pre==1)
#if NBODIES>4
			UpdateAccJerk_General<double>(&mPos[0], &mVel[0], &mAcc[0], &mJerk[0], NBODIES, &s_mass[0]);
#else
			UpdateAccJerk<double>(&mPos[0], &mVel[0], &mAcc[0], &mJerk[0], NBODIES, &s_mass[0]);
#endif
		else
		{
			doubleTofloat<nData>(sPos, mPos);
			doubleTofloat<nData>(sVel, mVel);
#if NBODIES>4
			UpdateAccJerk_General<float>(&sPos[0], &sVel[0], &sAcc[0], &sJerk[0], NBODIES, &s_mass[0]);
#else
			UpdateAccJerk<float>(&sPos[0], &sVel[0], &sAcc[0], &sJerk[0], NBODIES, &s_mass[0]);
#endif
			floatTodouble<nData>(mAcc,sAcc);
			floatTodouble<nData>(mJerk,sJerk);
		}

		//Correct(dt);
		correct(mPos,mVel,mAcc,mJerk, mPosOld,mVelOld,mAccOld,mJerkOld, dtby2, dtby6, dtby7, dt7by30, nData);

		T += dt;
	}


#if NBODIES>4
        for(unsigned int i=0;i<NBODIES;++i)
         {
           ens.x(sys,i)   =mPos[i*3  ];
           ens.y(sys,i)   =mPos[i*3+1];
           ens.z(sys,i)   =mPos[i*3+2];

           ens.vx(sys,i)  =mVel[i*3  ];
           ens.vy(sys,i)  =mVel[i*3+1];
           ens.vz(sys,i)  =mVel[i*3+2];
         }
#else
	ens.x(sys,0)=mPos[0];
	ens.y(sys,0)=mPos[1];
	ens.z(sys,0)=mPos[2];
	ens.x(sys,1)=mPos[3];
	ens.y(sys,1)=mPos[4];
	ens.z(sys,1)=mPos[5];
	ens.x(sys,2)=mPos[6];
	ens.y(sys,2)=mPos[7];
	ens.z(sys,2)=mPos[8];
#if NBODIES>3
	ens.x(sys,3)=mPos[9];
	ens.y(sys,3)=mPos[10];
	ens.z(sys,3)=mPos[11];
#endif

	ens.vx(sys,0)=mVel[0];
	ens.vy(sys,0)=mVel[1];
	ens.vz(sys,0)=mVel[2];
	ens.vx(sys,1)=mVel[3];
	ens.vy(sys,1)=mVel[4];
	ens.vz(sys,1)=mVel[5];
	ens.vx(sys,2)=mVel[6];
	ens.vy(sys,2)=mVel[7];
	ens.vz(sys,2)=mVel[8];
#if NBODIES>3
	ens.vx(sys,3)=mVel[9];
	ens.vy(sys,3)=mVel[10];
	ens.vz(sys,3)=mVel[11];
#endif
#endif




        ens.time(sys)=T;
}

void gpu_hermite_adap_integrator::integrate(gpu_ensemble &ens, double dT)
{
	// Upload the kernel parameters
	if(ens.last_integrator() != this)
	{
		ens.set_last_integrator(this);
		configure_grid(gridDim, threadsPerBlock, ens.nsys());

		hipMemcpyToSymbol(HIP_SYMBOL(gpu_hermite_adap_ens), &ens, sizeof(gpu_hermite_adap_ens));
		if(dT == 0.) { return; }
	}

	// execute the kernel
	switch(prec){
		// double precision
		case 1:
			gpu_hermite_adap_integrator_kernel<1><<<gridDim, threadsPerBlock>>>(dT, h);
			break;
		// signle precision
		case 2:
			gpu_hermite_adap_integrator_kernel<2><<<gridDim, threadsPerBlock>>>(dT, h);
			break;
		// mixed precision
		case 3:
			gpu_hermite_adap_integrator_kernel<3><<<gridDim, threadsPerBlock>>>(dT, h);
			break;
	}
	printf("%s\n", hipGetErrorString(hipGetLastError()));

}

